#include "hip/hip_runtime.h"
#include <opera.h>
#include <phantom.h>
#include <chrono>
#include <thread>
#include "data_q1.h"

using namespace cuTFHEpp;
using namespace opera;
using namespace std;

bool FAST_COMP = true;
bool CACHE_ENABLED = true;
bool NOCHECK = true;

/***
 * TPC-H Query 1 modified
  select
      l_returnflag,
      l_linestatus,
      sum(l_quantity) as sum_qty,
      sum(l_extendedprice) as sum_base_price,
      sum(l_extendedprice * (1 - l_discount)) as sum_disc_price,
      sum(l_extendedprice * (1 - l_discount) * (1 + l_tax)) as sum_charge,
      sum(l_discount) as sum_disc,
      count(*) as count_order
  from
      lineitem
  where
      l_shipdate <= date '1998-12-01' - interval '120' day
  group by
      l_returnflag,
      l_linestatus

    consider data encode by [yyyymmdd], 23 bits,
    group by $m$ types of l_returnflag, $n$ types of l_linestatus
*/

void predicate_evaluation(std::vector<std::vector<TLWELvl1>> &pred_cres,
                          std::vector<std::vector<uint32_t>> &pred_res,
                          std::vector<DataRecord> &data,
                          QueryRequest &query_data,
                          TFHESecretKey &sk,
                          TFHEEvalKey &ek,
                          size_t rows,
                          double &filter_time)
{
  cout << "copy eval key to GPU" << endl;
  Pointer<Context> context(ek);
  Context &ctx = context.get();
  cout << "eval key is copied to GPU" << endl;

  std::cout << "Predicate evaluation: " << std::endl;
  using P = Lvl2;

  // Encrypt database
  std::cout << "Encrypting Database..." << std::endl;
  std::vector<TLWELvl1> returnflag_ciphers(rows), linestatus_ciphers(rows);
  std::vector<TLWELvl2> ship_ciphers(rows);
  for (size_t i = 0; i < rows; i++) {
    auto row_data = data[i];
    returnflag_ciphers[i] = TFHEpp::tlweSymInt32Encrypt<Lvl1>(
        row_data.returnflag().value, Lvl1::α,
        pow(2., row_data.returnflag().scale_bits<Lvl1>()), sk.key.get<Lvl1>());
    linestatus_ciphers[i] = TFHEpp::tlweSymInt32Encrypt<Lvl1>(
        row_data.linestatus().value, Lvl1::α,
        pow(2., row_data.linestatus().scale_bits<Lvl1>()), sk.key.get<Lvl1>());
    ship_ciphers[i] = TFHEpp::tlweSymInt32Encrypt<Lvl2>(
        row_data.shipdate().value, Lvl2::α,
        pow(2., row_data.shipdate().scale_bits<Lvl2>()), sk.key.get<Lvl2>());
  }

  // Encrypt Predicate values
  std::cout << "Encrypting Predicate Values..." << std::endl;

  // check if the predicate is correct
  auto groupby_num = query_data.groupby_num();
  // pred_ship[rows]
  std::vector<uint32_t> pred_ship_res(rows, 0);
  // pred_group[rows][groupby_num]
  std::vector<std::vector<uint32_t>> pred_group_res(
      groupby_num, std::vector<uint32_t>(rows, 0));
  // pred_res[groupby_num][rows]
  pred_res.resize(groupby_num, std::vector<uint32_t>(rows, 1));
  pred_cres.resize(groupby_num, std::vector<TLWELvl1>(rows));

  // pred_part & pred_group
  for (size_t i = 0; i < rows; i++) {
    auto ship_record = query_data.shipdate();
    pred_ship_res[i] = !!(data[i].shipdate().value <= ship_record.value);
    for (size_t j = 0; j < groupby_num; j++) {
      auto index = query_data.group_index(j);
      pred_group_res[j][i] = (data[i].linestatus().value ==
                              query_data.linestatus()[index[0]].value) &&
                             (data[i].returnflag().value ==
                              query_data.returnflag()[index[1]].value);
    }
  }
  // pred_res
  for (size_t i = 0; i < groupby_num; i++) {
    for (size_t j = 0; j < rows; j++) {
      pred_res[i][j] = pred_group_res[i][j] & pred_ship_res[j];
    }
  }

  // Encrypt Predicates
  std::vector<TLWELvl2> pred_cipher_ship(rows);
  // pred_cipher_group
  std::vector<std::vector<TLWELvl1>> pred_cipher_linestatus;
  std::vector<std::vector<TLWELvl1>> pred_cipher_returnflag;
  // encrypt predicate part
  auto cipher_ship = TFHEpp::tlweSymInt32Encrypt<Lvl2>(
      query_data.shipdate().value, Lvl2::α,
      pow(2., data[0].shipdate().scale_bits<Lvl2>()), sk.key.get<Lvl2>());
  for (size_t i = 0; i < rows; i++) {
    pred_cipher_ship[i] = cipher_ship;
  }
  // encrypt group by part
  double linestatus_scale = pow(2., data[0].linestatus().scale_bits<Lvl1>());
  auto linestatus_group = query_data.linestatus();
  pred_cipher_linestatus.resize(linestatus_group.size());
  for (size_t i = 0; i < linestatus_group.size(); i++) {
    auto temp = TFHEpp::tlweSymInt32Encrypt<Lvl1>(linestatus_group[i].value, Lvl1::α,
                                          linestatus_scale, sk.key.get<Lvl1>());
    pred_cipher_linestatus[i].resize(rows);
    for (size_t j = 0; j < rows; j++)
      pred_cipher_linestatus[i][j] = temp;
  }
  double returnflag_scale = pow(2., data[0].returnflag().scale_bits<Lvl1>());
  auto returnflag_group = query_data.returnflag();
  pred_cipher_returnflag.resize(returnflag_group.size());
  for (size_t i = 0; i < returnflag_group.size(); i++) {
    auto temp = TFHEpp::tlweSymInt32Encrypt<Lvl1>(returnflag_group[i].value, Lvl1::α,
                                          returnflag_scale, sk.key.get<Lvl1>());
    pred_cipher_returnflag[i].resize(rows);
    for (size_t j = 0; j < rows; j++)
      pred_cipher_returnflag[i][j] = temp;
  }

  // Predicate Evaluation
  std::cout << "Start Predicate Evaluation..." << std::endl;
  std::vector<TLWELvl1> pred_ship_cres(rows);
  auto ship_bits = data[0].shipdate().bits;
  std::vector<std::vector<TLWELvl1>> pred_group_cres1(
      groupby_num, std::vector<TLWELvl1>(rows));
  std::vector<std::vector<TLWELvl1>> pred_group_cres2(
      groupby_num, std::vector<TLWELvl1>(rows));
  std::vector<std::vector<TLWELvl1>> pred_group_cres(
      groupby_num, std::vector<TLWELvl1>(rows));
  auto linestatus_bits = data[0].linestatus().bits;
  auto returnflag_bits = data[0].returnflag().bits;

  Pointer<BootstrappingData<Lvl02>> pt_bs_data(rows);
  auto &pt_bs_data_lvl1 = pt_bs_data.template safe_cast<BootstrappingData<Lvl01>>();

  std::vector<Pointer<cuTLWE<Lvl2>>> tlwe_data;
  tlwe_data.reserve(4);
  for (size_t i = 0; i < 4; ++i) tlwe_data.emplace_back(rows);

  Pointer<cuTLWE<Lvl2>> *pt_tlwe_data = tlwe_data.data();
  Pointer<cuTLWE<Lvl1>> *pt_tlwe_data_lvl1 = &pt_tlwe_data->template safe_cast<cuTLWE<Lvl1>>();

  filter_time = 0;

  HomComp<Lvl02, LE, LOGIC>(ctx, pt_bs_data, pt_tlwe_data,
      pred_ship_cres.data(), ship_ciphers.data(), pred_cipher_ship.data(),
      ship_bits, rows, filter_time);

  for (size_t j = 0; j < groupby_num; j++) {
    auto index = query_data.group_index(j);
    HomComp<Lvl01, EQ, LOGIC>(ctx, pt_bs_data_lvl1, pt_tlwe_data_lvl1,
        pred_group_cres1[j].data(), pred_cipher_linestatus[index[0]].data(), linestatus_ciphers.data(),
        linestatus_bits, rows, filter_time);
    HomComp<Lvl01, EQ, LOGIC>(ctx, pt_bs_data_lvl1, pt_tlwe_data_lvl1,
        pred_group_cres2[j].data(), pred_cipher_returnflag[index[1]].data(), returnflag_ciphers.data(),
        returnflag_bits, rows, filter_time);

    HomAND<LOGIC>(ctx, pt_bs_data_lvl1, pt_tlwe_data_lvl1,
        pred_group_cres[j].data(), pred_group_cres1[j].data(), pred_group_cres2[j].data(),
        rows, filter_time);
    HomAND<ARITHMETIC>(ctx, pt_bs_data_lvl1, pt_tlwe_data_lvl1,
        pred_cres[j].data(), pred_group_cres[j].data(), pred_ship_cres.data(),
        rows, filter_time);
  }

  // check the results
  if (!NOCHECK) {
    std::vector<std::vector<uint32_t>> pred_cres_de(groupby_num,
                                                    std::vector<uint32_t>(rows));
    std::vector<uint32_t> pred_ship_cres_de(rows);
    std::vector<std::vector<uint32_t>> pred_group_cres1_de(
        groupby_num, std::vector<uint32_t>(rows));
    std::vector<std::vector<uint32_t>> pred_group_cres2_de(
        groupby_num, std::vector<uint32_t>(rows));
    std::vector<std::vector<uint32_t>> pred_group_cres_de(
        groupby_num, std::vector<uint32_t>(rows));
    for (size_t i = 0; i < rows; i++) {
      pred_ship_cres_de[i] =
          TFHEpp::tlweSymDecrypt<Lvl1>(pred_ship_cres[i], sk.key.lvl1);
      for (size_t j = 0; j < groupby_num; j++) {
        pred_cres_de[j][i] = TFHEpp::tlweSymInt32Decrypt<Lvl1>(
            pred_cres[j][i], pow(2., 31), sk.key.get<Lvl1>());
        pred_group_cres1_de[j][i] =
            TFHEpp::tlweSymDecrypt<Lvl1>(pred_group_cres1[j][i], sk.key.lvl1);
        pred_group_cres2_de[j][i] =
            TFHEpp::tlweSymDecrypt<Lvl1>(pred_group_cres2[j][i], sk.key.lvl1);
        pred_group_cres_de[j][i] =
            TFHEpp::tlweSymDecrypt<Lvl1>(pred_group_cres[j][i], sk.key.lvl1);
      }
    }

    size_t error_time = 0;

    uint32_t rlwe_scale_bits = 29;
    for (size_t j = 0; j < groupby_num; j++)
      ari_rescale<Lvl10, Lvl01>(ctx, pt_bs_data_lvl1, pt_tlwe_data_lvl1,
          pred_cres[j].data(), pred_cres[j].data(), rlwe_scale_bits, rows);

    for (size_t i = 0; i < rows; i++)
      for (size_t j = 0; j < groupby_num; j++) {
        pred_cres_de[j][i] = TFHEpp::tlweSymInt32Decrypt<Lvl1>(
            pred_cres[j][i], pow(2., 29), sk.key.get<Lvl1>());
      }
    for (size_t i = 0; i < rows; i++)
      for (size_t j = 0; j < groupby_num; j++)
        error_time += (pred_cres_de[j][i] == pred_res[j][i]) ? 0 : 1;

    std::cout << "Predicate Error: " << error_time << std::endl;
  }

  std::cout << "Filter Time : " << filter_time << "ms" << std::endl;
}

void predicate_evaluation_cache(std::vector<std::vector<TLWELvl1>> &pred_cres,
                          std::vector<std::vector<uint32_t>> &pred_res,
                          std::vector<DataRecord> &data,
                          QueryRequest &query_data,
                          TFHESecretKey &sk,
                          TFHEEvalKey &ek,
                          CacheManager<Lvl1> &cm,
                          std::vector<std::vector<CacheFilter>> &filters,
                          std::vector<std::string> &filters_name,
                          std::vector<CacheMetadata<Lvl1::T>> &metas,
                          std::vector<std::vector<CacheFilter>> &gfilters,
                          std::vector<std::string> &gfilters_name,
                          std::vector<CacheMetadata<Lvl1::T>> &gmetas,
                          size_t rows,
                          double &filter_time,
                          double &tfhe_correction_time)
{
  cout << "copy eval key to GPU" << endl;
  Pointer<Context> context(ek);
  Context &ctx = context.get();
  cout << "eval key is copied to GPU" << endl;

  std::cout << "Predicate evaluation: " << std::endl;
  using P = Lvl2;

  // Encrypt database
  std::cout << "Encrypting Database..." << std::endl;
  std::vector<TLWELvl1> returnflag_ciphers(rows), linestatus_ciphers(rows);
  std::vector<TLWELvl2> ship_ciphers(rows);
  for (size_t i = 0; i < rows; i++) {
    auto row_data = data[i];
    returnflag_ciphers[i] = TFHEpp::tlweSymInt32Encrypt<Lvl1>(
        row_data.returnflag().value, Lvl1::α,
        pow(2., row_data.returnflag().scale_bits<Lvl1>()), sk.key.get<Lvl1>());
    linestatus_ciphers[i] = TFHEpp::tlweSymInt32Encrypt<Lvl1>(
        row_data.linestatus().value, Lvl1::α,
        pow(2., row_data.linestatus().scale_bits<Lvl1>()), sk.key.get<Lvl1>());
    ship_ciphers[i] = TFHEpp::tlweSymInt32Encrypt<Lvl2>(
        row_data.shipdate().value, Lvl2::α,
        pow(2., row_data.shipdate().scale_bits<Lvl2>()), sk.key.get<Lvl2>());
  }

  // Encrypt Predicate values
  std::cout << "Encrypting Predicate Values..." << std::endl;

  // check if the predicate is correct
  auto groupby_num = query_data.groupby_num();
  // pred_ship[rows]
  std::vector<uint32_t> pred_ship_res(rows, 0);
  // pred_group[rows][groupby_num]
  std::vector<std::vector<uint32_t>> pred_group_res(
      groupby_num, std::vector<uint32_t>(rows, 0));
  // pred_res[groupby_num][rows]
  pred_res.resize(groupby_num, std::vector<uint32_t>(rows, 1));
  pred_cres.resize(groupby_num, std::vector<TLWELvl1>(rows));

  // pred_part & pred_group
  for (size_t i = 0; i < rows; i++) {
    auto ship_record = query_data.shipdate();
    pred_ship_res[i] = !!(data[i].shipdate().value <= ship_record.value);
    for (size_t j = 0; j < groupby_num; j++) {
      auto index = query_data.group_index(j);
      pred_group_res[j][i] = (data[i].linestatus().value ==
                              query_data.linestatus()[index[0]].value) &&
                             (data[i].returnflag().value ==
                              query_data.returnflag()[index[1]].value);
    }
  }
  // pred_res
  for (size_t i = 0; i < groupby_num; i++) {
    for (size_t j = 0; j < rows; j++) {
      pred_res[i][j] = pred_group_res[i][j] & pred_ship_res[j];
    }
  }

  std::vector<Lvl1::T> data_shipdate;
  // ==== generate cache filters
  std::transform(data.begin(), data.end(), std::back_inserter(data_shipdate),
                 [](DataRecord &item) { return item.shipdate().value; });
  cm.generate(filters_name[0], data_shipdate, metas[0]);

  size_t i = 0;
  std::vector<Lvl1::T> data_linestatus, data_returnflag;
  std::transform(data.begin(), data.end(), std::back_inserter(data_linestatus),
                 [](DataRecord &item) { return item.linestatus().value; });
  std::transform(data.begin(), data.end(), std::back_inserter(data_returnflag),
                 [](DataRecord &item) { return item.returnflag().value; });
  for (size_t j = 0; j < gfilters[0].size(); ++i, ++j)
    cm.generate(gfilters_name[i], data_linestatus, gmetas[i]);
  for (size_t j = 0; j < gfilters[1].size(); ++i, ++j)
    cm.generate(gfilters_name[i], data_returnflag, gmetas[i]);
  // ==== end of cache filter generation

  // Encrypt Predicates
  std::vector<TLWELvl2> pred_cipher_ship(rows);
  // pred_cipher_group
  std::vector<std::vector<TLWELvl1>> pred_cipher_linestatus;
  std::vector<std::vector<TLWELvl1>> pred_cipher_returnflag;
  // encrypt predicate part
  auto cipher_ship = TFHEpp::tlweSymInt32Encrypt<Lvl2>(
      query_data.shipdate().value, Lvl2::α,
      pow(2., data[0].shipdate().scale_bits<Lvl2>()), sk.key.get<Lvl2>());
  for (size_t i = 0; i < rows; i++) {
    pred_cipher_ship[i] = cipher_ship;
  }
  // encrypt group by part
  double linestatus_scale = pow(2., data[0].linestatus().scale_bits<Lvl1>());
  auto linestatus_group = query_data.linestatus();
  pred_cipher_linestatus.resize(linestatus_group.size());
  for (size_t i = 0; i < linestatus_group.size(); i++) {
    auto temp = TFHEpp::tlweSymInt32Encrypt<Lvl1>(linestatus_group[i].value, Lvl1::α,
                                          linestatus_scale, sk.key.get<Lvl1>());
    pred_cipher_linestatus[i].resize(rows);
    for (size_t j = 0; j < rows; j++)
      pred_cipher_linestatus[i][j] = temp;
  }
  double returnflag_scale = pow(2., data[0].returnflag().scale_bits<Lvl1>());
  auto returnflag_group = query_data.returnflag();
  pred_cipher_returnflag.resize(returnflag_group.size());
  for (size_t i = 0; i < returnflag_group.size(); i++) {
    auto temp = TFHEpp::tlweSymInt32Encrypt<Lvl1>(returnflag_group[i].value, Lvl1::α,
                                          returnflag_scale, sk.key.get<Lvl1>());
    pred_cipher_returnflag[i].resize(rows);
    for (size_t j = 0; j < rows; j++)
      pred_cipher_returnflag[i][j] = temp;
  }

  // Predicate Evaluation
  std::cout << "Start Predicate Evaluation..." << std::endl;
  std::vector<TLWELvl1> pred_ship_cres(rows);
  auto ship_bits = data[0].shipdate().bits;
  std::vector<std::vector<TLWELvl1>> pred_group_cres1(
      groupby_num, std::vector<TLWELvl1>(rows));
  std::vector<std::vector<TLWELvl1>> pred_group_cres2(
      groupby_num, std::vector<TLWELvl1>(rows));
  std::vector<std::vector<TLWELvl1>> pred_group_cres(
      groupby_num, std::vector<TLWELvl1>(rows));
  auto linestatus_bits = data[0].linestatus().bits;
  auto returnflag_bits = data[0].returnflag().bits;

  // ==== find cache filters
  // predicates
  for (int i = 0; i < filters_name.size(); i++) {
    cm.find(filters_name[i], filters[i], metas[i]);
  }
  // groupby
  int col = 0, row = 0;
  for (int i = 0; i < gfilters_name.size(); i++, row++) {
    std::vector<CacheFilter> tmp;
    cm.find(gfilters_name[i], tmp, gmetas[i]);
    assert(tmp.size() < 2);
    if (!tmp.empty())
      gfilters[col][row] = tmp[0];
    else
      gfilters[col][row] = CacheFilter();
    // update col and row
    if (row == gfilters[col].size() - 1) {
      col++;
      row = -1;
    }
  }
  // ==== end of finding cache filters

  Pointer<BootstrappingData<Lvl02>> pt_bs_data(rows);
  auto &pt_bs_data_lvl1 = pt_bs_data.template safe_cast<BootstrappingData<Lvl01>>();

  std::vector<Pointer<cuTLWE<Lvl2>>> tlwe_data;
  tlwe_data.reserve(4);
  for (size_t i = 0; i < 4; ++i) tlwe_data.emplace_back(rows);

  Pointer<cuTLWE<Lvl2>> *pt_tlwe_data = tlwe_data.data();
  Pointer<cuTLWE<Lvl1>> *pt_tlwe_data_lvl1 = &pt_tlwe_data->template safe_cast<cuTLWE<Lvl1>>();

  filter_time = 0;
  tfhe_correction_time = 0;

  HomFastComp<Lvl02, LE, LOGIC>(ctx, pt_bs_data, pt_tlwe_data,
      pred_ship_cres.data(), ship_ciphers.data(), pred_cipher_ship.data(),
      ship_bits, metas[0].get_density(), rows, filter_time);

  tfhe_correction(ctx, filters[0], pt_bs_data_lvl1, pt_tlwe_data_lvl1, pred_ship_cres.data(),
      rows, tfhe_correction_time);

  std::vector<size_t> indices(gfilters.size(), 0);
  for (size_t j = 0; j < groupby_num; j++) {
    auto index = query_data.group_index(j);

    // group by - linestatus
    auto linestatus_filters = gfilters[0][indices[0]];
    auto linestatus_metas = gmetas[indices[0]];
    bool operated_linestatus = !!linestatus_metas.get_density();

    HomFastComp<Lvl01, EQ, LOGIC>(ctx, pt_bs_data_lvl1, pt_tlwe_data_lvl1,
        pred_group_cres1[j].data(), pred_cipher_linestatus[index[0]].data(), linestatus_ciphers.data(),
        linestatus_bits, linestatus_metas.get_density(), rows, filter_time);
    operated_linestatus = tfhe_correction(
        linestatus_filters, pt_tlwe_data_lvl1, pred_group_cres1[j].data(), rows, tfhe_correction_time)
      || operated_linestatus;

    // group by - returnflag
    auto base = gfilters[0].size();
    auto returnflag_filters = gfilters[1][indices[1]];
    auto returnflag_metas = gmetas[base + indices[1]];
    bool operated_returnflag = !!returnflag_metas.get_density();

    HomFastComp<Lvl01, EQ, LOGIC>(ctx, pt_bs_data_lvl1, pt_tlwe_data_lvl1,
        pred_group_cres2[j].data(), pred_cipher_returnflag[index[1]].data(), returnflag_ciphers.data(),
        returnflag_bits, returnflag_metas.get_density(), rows, filter_time);
    operated_returnflag = tfhe_correction(
        returnflag_filters, pt_tlwe_data_lvl1, pred_group_cres2[j].data(), rows, tfhe_correction_time)
      || operated_returnflag;

    if (operated_linestatus || operated_returnflag) {
      if (operated_linestatus && operated_returnflag) {
        HomAND<LOGIC>(ctx, pt_bs_data_lvl1, pt_tlwe_data_lvl1,
            pred_group_cres[j].data(), pred_group_cres1[j].data(), pred_group_cres2[j].data(),
            rows, filter_time);
      } else {
        pred_group_cres[j] = operated_linestatus ? pred_group_cres1[j]
                                                    : pred_group_cres2[j];
      }
      HomAND<ARITHMETIC>(ctx, pt_bs_data_lvl1, pt_tlwe_data_lvl1,
          pred_cres[j].data(), pred_group_cres[j].data(), pred_ship_cres.data(), rows, filter_time);
    } else {
      HomAND<ARITHMETIC>(ctx, pt_bs_data_lvl1, pt_tlwe_data_lvl1,
          pred_cres[j].data(), pred_ship_cres.data(), pred_ship_cres.data(), rows, filter_time);
    }

    // Move to next
    for (size_t k = gfilters.size(); k-- > 0;) {
      if (++indices[k] < gfilters[k].size()) {
        break;
      }
      indices[k] = 0;
    }
  }

  // check the results
  if (!NOCHECK) {
    std::vector<std::vector<uint32_t>> pred_cres_de(groupby_num,
                                                    std::vector<uint32_t>(rows));
    std::vector<uint32_t> pred_ship_cres_de(rows);
    std::vector<std::vector<uint32_t>> pred_group_cres1_de(
        groupby_num, std::vector<uint32_t>(rows));
    std::vector<std::vector<uint32_t>> pred_group_cres2_de(
        groupby_num, std::vector<uint32_t>(rows));
    std::vector<std::vector<uint32_t>> pred_group_cres_de(
        groupby_num, std::vector<uint32_t>(rows));
    for (size_t i = 0; i < rows; i++) {
      pred_ship_cres_de[i] =
          TFHEpp::tlweSymDecrypt<Lvl1>(pred_ship_cres[i], sk.key.lvl1);
      for (size_t j = 0; j < groupby_num; j++) {
        pred_cres_de[j][i] = TFHEpp::tlweSymInt32Decrypt<Lvl1>(
            pred_cres[j][i], pow(2., 31), sk.key.get<Lvl1>());
        pred_group_cres1_de[j][i] =
            TFHEpp::tlweSymDecrypt<Lvl1>(pred_group_cres1[j][i], sk.key.lvl1);
        pred_group_cres2_de[j][i] =
            TFHEpp::tlweSymDecrypt<Lvl1>(pred_group_cres2[j][i], sk.key.lvl1);
        pred_group_cres_de[j][i] =
            TFHEpp::tlweSymDecrypt<Lvl1>(pred_group_cres[j][i], sk.key.lvl1);
      }
    }

    size_t error_time = 0;

    uint32_t rlwe_scale_bits = 29;
    for (size_t j = 0; j < groupby_num; j++)
      ari_rescale<Lvl10, Lvl01>(ctx, pt_bs_data_lvl1, pt_tlwe_data_lvl1,
          pred_cres[j].data(), pred_cres[j].data(), rlwe_scale_bits, rows);

    for (size_t i = 0; i < rows; i++)
      for (size_t j = 0; j < groupby_num; j++) {
        pred_cres_de[j][i] = TFHEpp::tlweSymInt32Decrypt<Lvl1>(
            pred_cres[j][i], pow(2., 29), sk.key.get<Lvl1>());
      }
    for (size_t i = 0; i < rows; i++)
      for (size_t j = 0; j < groupby_num; j++)
        error_time += (pred_cres_de[j][i] == pred_res[j][i]) ? 0 : 1;
    std::cout << "Predicate Error: " << error_time << std::endl;
  }

  std::cout << "[PHC] " << filter_time << "ms" << std::endl;
  std::cout << "[TFHE Correction] " << tfhe_correction_time << "ms"
            << std::endl;
  filter_time += tfhe_correction_time;
  std::cout << "[Evaluation] " << filter_time << "ms" << std::endl;
}

void aggregation(std::vector<PhantomCiphertext> &result,
                 std::vector<std::vector<uint32_t>> &pred_res,
                 std::vector<DataRecord> &data, size_t rows, PhantomRLWE &rlwe,
                 double &aggregation_time) {
  std::cout << "Aggregation :" << std::endl;
  size_t groupby_num = result.size();

  // Table for data, ciphertext, and aggregation results
  struct DataPack {
    std::vector<double> &data;
    PhantomCiphertext &cipher;
    std::vector<PhantomCiphertext> &sum;
  };

  // Filter result * data
  // original data
  std::vector<double> quantity_data(rows), extendedprice_data(rows),
      extendedprice_discount_data(rows), discount_tax_data(rows),
      discount_data(rows);
  // packed ciphertext
  PhantomCiphertext quantity_cipher, extendedprice_cipher,
      extendedprice_discount_cipher, discount_tax_cipher, discount_cipher;
  // sum result ciphertext
  std::vector<PhantomCiphertext> sum_qty(groupby_num),
      sum_base_price(groupby_num), sum_disc_price(groupby_num),
      sum_charge(groupby_num), sum_disc(groupby_num);
  std::vector<DataPack> table = {
      {quantity_data, quantity_cipher, sum_qty},
      {extendedprice_data, extendedprice_cipher, sum_base_price},
      {extendedprice_discount_data, extendedprice_discount_cipher,
       sum_disc_price},
      {discount_tax_data, discount_tax_cipher, sum_charge},
      {discount_data, discount_cipher, sum_disc}};

  for (size_t i = 0; i < rows; i++) {
    quantity_data[i] = data[i].quantity().value;
    extendedprice_data[i] = data[i].extendedprice().value;
    extendedprice_discount_data[i] =
        data[i].extendedprice().value * (1 - data[i].discount().value);
    discount_tax_data[i] = data[i].extendedprice().value *
                           (1 - data[i].discount().value) *
                           (1 + data[i].tax().value);
    discount_data[i] = data[i].discount().value;
  }

  // convert data to ciphertext
  PhantomPlaintext t_plain;
  double qd =
      rlwe.parms.coeff_modulus()[result[0].coeff_modulus_size_ - 1].value();
  for (auto [_data_plaintext, _data_cipher, _sum_cipher] : table) {
    pack_encode(*rlwe.context, _data_plaintext, qd, t_plain, *rlwe.ckks_encoder);
    rlwe.secret_key->encrypt_symmetric(*rlwe.context, t_plain, _data_cipher, false);
  }


  std::cout << "Aggregating quantity, prices and discount .." << std::endl;
  // filtering the data
  std::chrono::system_clock::time_point start, end;
  start = std::chrono::system_clock::now();
  for (size_t i = 0; i < groupby_num; ++i) {
    for (auto [_data_plaintext, _data_cipher, _sum_cipher] : table) {
      multiply_and_relinearize(*rlwe.context, result[i], _data_cipher, _sum_cipher[i],
                                     *rlwe.relin_keys);
      rescale_to_next_inplace(*rlwe.context, _sum_cipher[i]);
    }
  }
  hipDeviceSynchronize();
  end = std::chrono::system_clock::now();
  aggregation_time =
      std::chrono::duration_cast<std::chrono::nanoseconds>(end - start)
          .count();

  // sum to aggregation
  int logrow = log2(rows);
  PhantomCiphertext temp;
  start = std::chrono::system_clock::now();
  for (size_t i = 0; i < groupby_num; ++i) {
    for (size_t j = 0; j < logrow; j++) {
      size_t step = 1 << (logrow - j - 1);
      for (auto [_data_plaintext, _data_cipher, _sum_cipher] : table) {
        temp = _sum_cipher[i];
        rotate_vector_inplace(*rlwe.context, temp, step, *rlwe.galois_keys);
        add_inplace(*rlwe.context, _sum_cipher[i], temp);
      }
    }
  }
  end = std::chrono::system_clock::now();
  aggregation_time +=
      std::chrono::duration_cast<std::chrono::nanoseconds>(end - start)
          .count();
  aggregation_time /= 1000000;
  std::cout << "Aggregation Time: " << aggregation_time << " ms" << std::endl;
  // Decrypt and check the result
  if (!NOCHECK) {
    std::vector<double> agg_result(rows);
    for (size_t i = 0; i < groupby_num; ++i) {
      for (auto [_data_plaintext, _data_cipher, _sum_cipher] : table) {
        rlwe.secret_key->decrypt(*rlwe.context, _sum_cipher[i], t_plain);
        pack_decode(*rlwe.context, agg_result, t_plain, *rlwe.ckks_encoder);
        double plain_result = 0;
        for (size_t j = 0; j < rows; j++) {
          plain_result += _data_plaintext[j] * pred_res[i][j];
        }
        cout << "Plain_result/Encrypted query result: " << plain_result << "/"
             << agg_result[0] << endl;
      }
    }
  }
}

void query_evaluation(TFHESecretKey &sk, TFHEEvalKey &ek, size_t rows, std::vector<double> &time)
{
  cout << "===== Query Evaluation: " << rows << " rows =====" << endl;
    // Generate database
  vector<DataRecord> data(rows);
  QueryRequest query_data;
  int returnflag_size = 2, linestatus_size = 3;
  for (size_t i = 0; i < rows; i++) {
    data[i].init(returnflag_size, linestatus_size);
  }
  query_data.init(returnflag_size, linestatus_size);
  PhantomRLWE rlwe(rows);

  if (!CACHE_ENABLED) {
    double filter_time, conversion_time, aggregation_time;
    std::vector<std::vector<TLWELvl1>> pred_cres;
    std::vector<std::vector<uint32_t>> pred_res;
    std::vector<PhantomCiphertext> results;

    predicate_evaluation(pred_cres, pred_res, data, query_data, sk, ek, rows, filter_time);
    rlwe.genLWE2RLWEGaloisKeys();
    conversion(results, pred_cres, pred_res, rlwe, sk, conversion_time, NOCHECK);
    rlwe.genGaloisKeys();
    aggregation(results, pred_res, data, rows, rlwe, aggregation_time);
    cout << "End-to-End Time: "
         << (filter_time + conversion_time + aggregation_time) / 1000 << " s"
         << endl;

    time.push_back(rows);
    time.push_back(filter_time/1000);
    time.push_back(conversion_time/1000);
    time.push_back(aggregation_time/1000);
    time.push_back((filter_time+conversion_time+aggregation_time)/1000);
    return;
  }

  using T = Lvl1::T;
  CacheManager<Lvl1> cm(&sk, &ek, &rlwe, FAST_COMP);

  std::vector<std::string> filters_name = {"shipdate"};
  std::vector<std::vector<CacheFilter>> filters(filters_name.size());
  std::vector<CacheMetadata<T>> metas = {
      CacheMetadata<T>(CompLogic::LE, (T)query_data.shipdate().value)};

  std::vector<std::string> gfilters_name;
  std::vector<std::vector<CacheFilter>> gfilters(2);
  std::vector<CacheMetadata<T>> gmetas;
  gfilters[0] = std::vector<CacheFilter>(linestatus_size);
  for (size_t i = 0; i < linestatus_size; ++i) {
    gfilters_name.push_back("linestatus");
    gmetas.push_back(CacheMetadata<T>(CompLogic::EQ,
                                 (T)query_data.linestatus()[i].value));
  };
  gfilters[1] = std::vector<CacheFilter>(returnflag_size);
  for (size_t i = 0; i < returnflag_size; ++i) {
    gfilters_name.push_back("returnflag");
    gmetas.push_back(CacheMetadata<T>(CompLogic::EQ,
                                 (T)query_data.returnflag()[i].value));
  };

  double filter_time, conversion_time, tfhe_correction_time, ckks_correction_time, aggregation_time;
  std::vector<std::vector<TLWELvl1>> pred_cres;
  std::vector<std::vector<uint32_t>> pred_res;
  std::vector<PhantomCiphertext> results;

  predicate_evaluation_cache(pred_cres, pred_res, data, query_data, sk, ek,
      cm, filters, filters_name, metas, gfilters, gfilters_name, gmetas, rows, filter_time, tfhe_correction_time);
  rlwe.genLWE2RLWEGaloisKeys();
  conversion(results, pred_cres, pred_res, rlwe, sk, conversion_time, NOCHECK);
  // phantom::util::global_pool()->Release_useless();
  rlwe.genGaloisKeys();
  filter_correction(results, pred_res, rlwe, filters, gfilters,
                  ckks_correction_time, NOCHECK);
  aggregation(results, pred_res, data, rows, rlwe, aggregation_time);
  cout << "End-to-End Time: "
       << (filter_time + tfhe_correction_time + conversion_time + ckks_correction_time + aggregation_time) / 1000 << " s"
       << endl;
  time.push_back(rows);
  time.push_back((filter_time+tfhe_correction_time+ckks_correction_time)/1000);
  time.push_back(filter_time/1000);
  time.push_back(tfhe_correction_time/1000);
  time.push_back(ckks_correction_time/1000);
  time.push_back(conversion_time/1000);
  time.push_back(aggregation_time/1000);
  time.push_back((filter_time+tfhe_correction_time+ckks_correction_time+conversion_time+aggregation_time)/1000);
}

int main(int argc, char** argv)
{
  argparse::ArgumentParser program("tpch_q1");

  program.add_argument("--nofastcomp")
    .help("disable fastcomp")
    .default_value(false)
    .implicit_value(true);

  program.add_argument("--nocache")
    .help("disable cache")
    .default_value(false)
    .implicit_value(true);

  program.add_argument("--check")
    .help("check result")
    .default_value(false)
    .implicit_value(true);

  program.add_argument("-o", "--output")
    .help("output file")
    .default_value(std::string(""));

  program.add_argument("--rows")
    .help("number of rows")
    .nargs(1,10)
    .scan<'i', int>();

  program.add_argument("-d", "--device")
    .help("device id")
    .default_value(0)
    .scan<'i', int>();

  try {
    program.parse_args(argc, argv);
  } catch (const std::exception &err) {
    std::cerr << err.what() << std::endl;
    std::cerr << program;
    return 1;
  }

  FAST_COMP = program["--nofastcomp"] == false;
  CACHE_ENABLED = program["--nocache"] == false;
  NOCHECK = program["--check"] == false;
  auto output = program.get<std::string>("-o");
  auto rows = program.get<std::vector<int>>("--rows");
  auto device = program.get<int>("-d");
  int n = rows.size();

  hipSetDevice(device);
  TFHESecretKey sk;
  TFHEEvalKey ek;

  load_keys<BootstrappingKeyFFTLvl01, BootstrappingKeyFFTLvl02,
    KeySwitchingKeyLvl10, KeySwitchingKeyLvl20, KeySwitchingKeyLvl21>(sk, ek);

  std::vector<std::vector<double>> time(n, std::vector<double>());
  for (int i = 0; i < n; i++) {
    query_evaluation(sk, ek, rows[i], time[i]);
    phantom::util::global_pool()->Release();
  }

  string output_head = CACHE_ENABLED ?
    "rows,fhc,phc,lwe_correct,rlwe_correct,packing,aggregation,end2end" :
    "rows,filter,packing,aggregation,end2end";

  if (output.empty()) {
    cout << "--------------------------------" << endl;
    cout << output_head << endl;
    for (size_t i = 0; i < time.size(); i++) {
      for (size_t j = 0; j < time[i].size(); j++) {
        cout << time[i][j] << ",";
      }
      cout << endl;
    }
  }
  else {
    ofstream ofs(output);
    ofs << output_head << endl;
    for (size_t i = 0; i < time.size(); i++) {
      for (size_t j = 0; j < time[i].size(); j++) {
        ofs << time[i][j] << ",";
      }
      ofs << endl;
    }
  }
}

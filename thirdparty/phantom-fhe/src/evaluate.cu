#include "hip/hip_runtime.h"
#include "evaluate.h"

#include "mempool.cuh"
#include "rns_bconv.cuh"
#include "scalingvariant.cuh"
#include "util.cuh"

using namespace std;
using namespace phantom;
using namespace phantom::util;

static void negate_internal(const PhantomContext& context, PhantomCiphertext& encrypted, const hipStream_t stream) {
    // Extract encryption parameters.
    auto& context_data = context.get_context_data(encrypted.chain_index());
    auto& parms = context_data.parms();
    auto& coeff_modulus = parms.coeff_modulus();
    const auto coeff_mod_size = coeff_modulus.size();
    const auto poly_degree = parms.poly_modulus_degree();
    const auto base_rns = context.gpu_rns_tables().modulus();
    const auto rns_coeff_count = poly_degree * coeff_mod_size;

    uint64_t gridDimGlb = rns_coeff_count / blockDimGlb.x;
    for (size_t i = 0; i < encrypted.size(); i++) {
        negate_rns_poly<<<gridDimGlb, blockDimGlb, 0, stream>>>(encrypted.data() + i * rns_coeff_count, base_rns,
                                                                encrypted.data() + i * rns_coeff_count,
                                                                poly_degree,
                                                                coeff_mod_size);
    }
}

void negate_inplace(const PhantomContext& context, PhantomCiphertext& encrypted) {
    negate_internal(context, encrypted, hipStreamLegacy);
}

void negate_inplace_async(const PhantomContext& context, PhantomCiphertext& encrypted, const hipStream_t stream) {
    negate_internal(context, encrypted, stream);
}

/**
 * Adds two ciphertexts. This function adds together encrypted1 and encrypted2 and stores the result in encrypted1.
 * @param[in] encrypted1 The first ciphertext to add
 * @param[in] encrypted2 The second ciphertext to add
 */
void add_inplace(const PhantomContext& context, PhantomCiphertext& encrypted1, const PhantomCiphertext& encrypted2) {
    if (encrypted1.chain_index() != encrypted2.chain_index()) {
        throw std::invalid_argument("encrypted1 and encrypted2 parameter mismatch");
    }
    if (encrypted1.is_ntt_form() != encrypted2.is_ntt_form()) {
        throw std::invalid_argument("NTT form mismatch");
    }
    if (!phantom::util::are_close(encrypted1.scale(), encrypted2.scale())) {
        throw std::invalid_argument("scale mismatch");
    }
    if (encrypted1.size_ != encrypted2.size_) {
        throw std::invalid_argument("poly number mismatch");
    }

    // Extract encryption parameters.
    auto& context_data = context.get_context_data(encrypted1.chain_index());
    auto& parms = context_data.parms();
    auto& coeff_modulus = parms.coeff_modulus();
    auto& plain_modulus = parms.plain_modulus();
    auto coeff_modulus_size = coeff_modulus.size();
    auto poly_degree = context.gpu_rns_tables().n();
    auto base_rns = context.gpu_rns_tables().modulus();
    auto rns_coeff_count = poly_degree * coeff_modulus_size;
    size_t encrypted1_size = encrypted1.size();
    size_t encrypted2_size = encrypted2.size();
    size_t max_size = max(encrypted1_size, encrypted2_size);
    size_t min_size = min(encrypted1_size, encrypted2_size);

    uint64_t gridDimGlb = rns_coeff_count / blockDimGlb.x;

    if (encrypted1.correction_factor() != encrypted2.correction_factor()) {
        // Balance correction factors and multiply by scalars before addition in BGV
        auto factors = balance_correction_factors(encrypted1.correction_factor(), encrypted2.correction_factor(),
                                                  plain_modulus);
        for (size_t i = 0; i < encrypted1.size(); i++) {
            multiply_scalar_rns_poly<<<gridDimGlb, blockDimGlb>>>(
                encrypted1.data() + i * rns_coeff_count, get<1>(factors), base_rns,
                encrypted1.data() + i * rns_coeff_count, poly_degree, coeff_modulus_size);
        }

        PhantomCiphertext encrypted2_copy(context);
        encrypted2_copy = encrypted2;
        for (size_t i = 0; i < encrypted2.size(); i++) {
            multiply_scalar_rns_poly<<<gridDimGlb, blockDimGlb>>>(
                encrypted2_copy.data() + i * rns_coeff_count, get<2>(factors), base_rns,
                encrypted2_copy.data() + i * rns_coeff_count, poly_degree, coeff_modulus_size);
        }

        // Set new correction factor
        encrypted1.correction_factor() = get<0>(factors);
        encrypted2_copy.correction_factor() = get<0>(factors);

        add_inplace(context, encrypted1, encrypted2_copy);
    } else {
        // Prepare destination
        encrypted1.resize(context, context_data.chain_index(), max_size);
        for (size_t i = 0; i < min_size; i++) {
            add_rns_poly<<<gridDimGlb, blockDimGlb>>>(
                encrypted1.data() + i * rns_coeff_count, encrypted2.data() + i * rns_coeff_count, base_rns,
                encrypted1.data() + i * rns_coeff_count, poly_degree, coeff_modulus_size);
        }
        if (encrypted1_size < encrypted2_size) {
            CUDA_CHECK(hipMemcpy(encrypted1.data() + min_size * rns_coeff_count,
                encrypted2.data() + min_size * rns_coeff_count,
                (encrypted2_size - encrypted1_size) * rns_coeff_count * sizeof(uint64_t),
                hipMemcpyDeviceToDevice));
        }
    }
}

// TODO: fixme
void add_many(const PhantomContext& context, const vector<PhantomCiphertext>& encrypteds,
              PhantomCiphertext& destination) {
    if (encrypteds.empty()) {
        throw std::invalid_argument("encrypteds cannot be empty");
    }
    for (size_t i = 0; i < encrypteds.size(); i++) {
        if (&encrypteds[i] == &destination) {
            throw std::invalid_argument("encrypteds must be different from destination");
        }
        if (encrypteds[0].chain_index() != encrypteds[i].chain_index()) {
            throw invalid_argument("encrypteds parameter mismatch");
        }
        if (encrypteds[0].is_ntt_form() != encrypteds[i].is_ntt_form()) {
            throw std::invalid_argument("NTT form mismatch");
        }
        if (encrypteds[0].scale() != encrypteds[i].scale()) {
            throw std::invalid_argument("scale mismatch");
        }
        if (encrypteds[0].size() != encrypteds[i].size()) {
            throw std::invalid_argument("poly number mismatch");
        }
    }

    auto& context_data = context.get_context_data(encrypteds[0].chain_index());
    auto& parms = context_data.parms();
    auto& coeff_modulus = parms.coeff_modulus();
    auto coeff_mod_size = coeff_modulus.size();
    auto poly_degree = parms.poly_modulus_degree();
    auto poly_num = encrypteds[0].size();
    auto base_rns = context.gpu_rns_tables().modulus();
    // reduction_threshold = 2 ^ (64 - max modulus bits)
    // max modulus bits = static_cast<uint64_t>(log2(coeff_modulus.front().value())) + 1
    auto reduction_threshold =
            (1 << (bits_per_uint64 - static_cast<uint64_t>(log2(coeff_modulus.front().value())) - 1)) - 1;

    destination.resize(context, encrypteds[0].chain_index(), encrypteds[0].size());
    destination.is_ntt_form() = encrypteds[0].is_ntt_form();
    destination.scale() = encrypteds[0].scale();

    if (parms.scheme() == phantom::scheme_type::bgv) // TODO: any optimizations?
    {
        CUDA_CHECK(hipMemcpy(destination.data(), encrypteds[0].data(),
            poly_degree * coeff_mod_size * encrypteds[0].size() * sizeof(uint64_t),
            hipMemcpyDeviceToDevice));
        for (size_t i = 1; i < encrypteds.size(); i++) {
            add_inplace(context, destination, encrypteds[i]);
        }
    } else {
        Pointer<uint64_t *> enc_device_ptr;
        enc_device_ptr.acquire(allocate<uint64_t *>(global_pool(), encrypteds.size()));
        uint64_t* enc_host_ptr[encrypteds.size()];

        for (size_t i = 0; i < encrypteds.size(); i++) {
            enc_host_ptr[i] = encrypteds[i].data();
        }
        CUDA_CHECK(hipMemcpy(enc_device_ptr.get(), enc_host_ptr, sizeof(uint64_t *) * encrypteds.size(),
            hipMemcpyHostToDevice));

        uint64_t gridDimGlb = poly_degree * coeff_mod_size / blockDimGlb.x;
        for (size_t i = 0; i < poly_num; i++) {
            add_many_rns_poly<<<gridDimGlb, blockDimGlb>>>(enc_device_ptr.get(), encrypteds.size(), base_rns,
                                                           destination.data(), i, poly_degree, coeff_mod_size,
                                                           reduction_threshold);
        }
    }
}

void sub_inplace(const PhantomContext& context, PhantomCiphertext& encrypted1, const PhantomCiphertext& encrypted2,
                 const bool& negate) {
    if (encrypted1.parms_id() != encrypted2.parms_id()) {
        throw invalid_argument("encrypted1 and encrypted2 parameter mismatch");
    }
    if (encrypted1.chain_index() != encrypted2.chain_index())
        throw std::invalid_argument("encrypted1 and encrypted2 parameter mismatch");
    if (encrypted1.is_ntt_form() != encrypted2.is_ntt_form())
        throw std::invalid_argument("NTT form mismatch");
    if (encrypted1.scale() != encrypted2.scale())
        throw std::invalid_argument("scale mismatch");
    if (encrypted1.size_ != encrypted2.size_)
        throw std::invalid_argument("poly number mismatch");

    // Extract encryption parameters.
    auto& context_data = context.get_context_data(encrypted1.chain_index());
    auto& parms = context_data.parms();
    auto& coeff_modulus = parms.coeff_modulus();
    auto& plain_modulus = parms.plain_modulus();
    auto coeff_modulus_size = coeff_modulus.size();
    auto poly_degree = parms.poly_modulus_degree();
    auto base_rns = context.gpu_rns_tables().modulus();
    auto rns_coeff_count = poly_degree * coeff_modulus_size;
    size_t encrypted1_size = encrypted1.size();
    size_t encrypted2_size = encrypted2.size();
    size_t max_count = max(encrypted1_size, encrypted2_size);
    size_t min_count = min(encrypted1_size, encrypted2_size);

    uint64_t gridDimGlb = rns_coeff_count / blockDimGlb.x;

    if (encrypted1.correction_factor() != encrypted2.correction_factor()) {
        // Balance correction factors and multiply by scalars before addition in BGV
        auto factors = balance_correction_factors(encrypted1.correction_factor(), encrypted2.correction_factor(),
                                                  plain_modulus);
        for (size_t i = 0; i < encrypted1.size(); i++) {
            multiply_scalar_rns_poly<<<gridDimGlb, blockDimGlb>>>(
                encrypted1.data() + i * rns_coeff_count, get<1>(factors), base_rns,
                encrypted1.data() + i * rns_coeff_count, poly_degree, coeff_modulus_size);
        }

        PhantomCiphertext encrypted2_copy(context);
        encrypted2_copy = encrypted2;
        for (size_t i = 0; i < encrypted2.size(); i++) {
            multiply_scalar_rns_poly<<<gridDimGlb, blockDimGlb>>>(
                encrypted2_copy.data() + i * rns_coeff_count, get<2>(factors), base_rns,
                encrypted2_copy.data() + i * rns_coeff_count, poly_degree, coeff_modulus_size);
        }

        // Set new correction factor
        encrypted1.correction_factor() = get<0>(factors);
        encrypted2_copy.correction_factor() = get<0>(factors);

        sub_inplace(context, encrypted1, encrypted2_copy, negate);
    } else {
        if (negate) {
            for (size_t i = 0; i < encrypted1.size(); i++) {
                sub_rns_poly<<<gridDimGlb, blockDimGlb>>>(
                    encrypted2.data() + i * rns_coeff_count, encrypted1.data() + i * rns_coeff_count, base_rns,
                    encrypted1.data() + i * rns_coeff_count, poly_degree, coeff_modulus_size);
            }
        } else {
            for (size_t i = 0; i < encrypted1.size(); i++) {
                sub_rns_poly<<<gridDimGlb, blockDimGlb>>>(
                    encrypted1.data() + i * rns_coeff_count, encrypted2.data() + i * rns_coeff_count, base_rns,
                    encrypted1.data() + i * rns_coeff_count, poly_degree, coeff_modulus_size);
            }
        }
    }
}

void bgv_ckks_multiply(const PhantomContext& context, PhantomCiphertext& encrypted1,
                       const PhantomCiphertext& encrypted2) {
    if (!(encrypted1.is_ntt_form() && encrypted2.is_ntt_form()))
        throw invalid_argument("encrypted1 and encrypted2 must be in NTT form");

    if (encrypted1.chain_index() != encrypted2.chain_index())
        throw invalid_argument("encrypted1 and encrypted2 parameter mismatch");

    // Extract encryption parameters.
    auto& context_data = context.get_context_data(encrypted1.chain_index());
    auto& parms = context_data.parms();
    auto base_rns = context.gpu_rns_tables().modulus();
    size_t coeff_mod_size = parms.coeff_modulus().size();
    size_t poly_degree = parms.poly_modulus_degree();
    uint32_t encrypted1_size = encrypted1.size();
    uint32_t encrypted2_size = encrypted2.size();

    // Determine destination.size()
    // Default is 3 (c_0, c_1, c_2)
    uint32_t dest_size = encrypted1_size + encrypted2_size - 1;

    // Size check
    // Prepare destination
    encrypted1.resize(context, encrypted1.chain_index(), dest_size);

    uint64_t gridDimGlb = poly_degree * coeff_mod_size / blockDimGlb.x;

    if (dest_size == 3) {
        if (&encrypted1 == &encrypted2) {
            // square
            tensor_square_2x2_rns_poly<<<gridDimGlb, blockDimGlb>>>(encrypted1.data(), base_rns, encrypted1.data(),
                                                                    poly_degree, coeff_mod_size);
        } else {
            // standard multiply
            tensor_prod_2x2_rns_poly<<<gridDimGlb, blockDimGlb>>>(encrypted1.data(), encrypted2.data(), base_rns,
                                                                  encrypted1.data(), poly_degree, coeff_mod_size);
        }
    } else {
        tensor_prod_mxn_rns_poly<<<gridDimGlb, blockDimGlb>>>(encrypted1.data(), encrypted1_size, encrypted2.data(),
                                                              encrypted2_size, base_rns, encrypted1.data(), dest_size,
                                                              poly_degree, coeff_mod_size);
    }

    // CKKS needs to do scaling
    if (parms.scheme() == scheme_type::ckks)
        encrypted1.scale() *= encrypted2.scale();

    // BGV needs to update correction factor
    if (parms.scheme() == scheme_type::bgv)
        encrypted1.correction_factor() = multiply_uint_mod(encrypted1.correction_factor(),
                                                           encrypted2.correction_factor(), parms.plain_modulus());
}

// Perform BEHZ steps (1)-(3) for PhantomCiphertext
// (1) Lift encrypted (initially in base q) to an extended base q U Bsk U {m_tilde}
// (2) Remove extra multiples of q from the results with Montgomery reduction, switching base to q U Bsk
// (3) Transform the data to NTT form
// @notice: temp is used to avoid memory malloc in sm_mrq
void BEHZ_mul_1(const PhantomContext& context, const PhantomCiphertext& encrypted, uint64_t* encrypted_q,
                uint64_t* encrypted_Bsk) {
    auto& context_data = context.get_context_data(encrypted.chain_index());
    auto& parms = context_data.parms();
    auto poly_degree = parms.poly_modulus_degree();
    auto& rns_tool = context.get_context_data(encrypted.chain_index()).gpu_rns_tool();

    size_t base_q_size = rns_tool.base_Ql().size();
    size_t base_Bsk_size = rns_tool.base_Bsk().size();
    size_t base_Bsk_m_tilde_size = rns_tool.base_Bsk_m_tilde().size();

    size_t q_coeff_count = poly_degree * base_q_size;
    size_t bsk_coeff_count = poly_degree * base_Bsk_size;

    Pointer<uint64_t> temp_base_Bsk_m_tilde;
    temp_base_Bsk_m_tilde.acquire(allocate<uint64_t>(global_pool(), poly_degree * base_Bsk_m_tilde_size));

    CUDA_CHECK(hipMemcpy(encrypted_q, encrypted.data(), encrypted.size() * q_coeff_count * sizeof(uint64_t),
        hipMemcpyDeviceToDevice));

    for (size_t i = 0; i < encrypted.size(); i++) {
        uint64_t* encrypted_ptr = encrypted.data() + i * q_coeff_count;
        uint64_t* encrypted_q_ptr = encrypted_q + i * q_coeff_count;
        uint64_t* encrypted_bsk_ptr = encrypted_Bsk + i * bsk_coeff_count;
        // NTT forward
        nwt_2d_radix8_forward_inplace(encrypted_q_ptr, context.gpu_rns_tables(), base_q_size, 0);
        // (1) Convert from base q to base Bsk U {m_tilde}
        rns_tool.fastbconv_m_tilde(temp_base_Bsk_m_tilde.get(), encrypted_ptr);
        // (2) Reduce q-overflows in with Montgomery reduction, switching base to Bsk
        rns_tool.sm_mrq(encrypted_bsk_ptr, temp_base_Bsk_m_tilde.get());
        // NTT forward
        nwt_2d_radix8_forward_inplace_include_temp_mod(encrypted_bsk_ptr, rns_tool.gpu_Bsk_tables(), base_Bsk_size, 0,
                                                       rns_tool.gpu_Bsk_tables().size());
    }
    temp_base_Bsk_m_tilde.release();
}

// encrypted1 = encrypted1 * encrypted2
// (c0, c1) * (c0', c1') = (c0*c0', c0'c1+c0c1', c1c1')
// BEHZ RNS multiplication, which completes the multiplication in RNS form.
// (1) Lift encrypted1 and encrypted2 (initially in base q) to an extended base q U Bsk U {m_tilde}
// (2) Remove extra multiples of q from the results with Montgomery reduction, switching base to q U Bsk
// (3) Transform the data to NTT form
// (4) Compute the ciphertext polynomial product using dyadic multiplication
// (5) Transform the data back from NTT form
// (6) Multiply the result by t (plain_modulus)
// (7) Scale the result by q using a divide-and-floor algorithm, switching base to Bsk
// (8) Use Shenoy-Kumaresan method to convert the result to base q
void bfv_multiply_behz(const PhantomContext& context, PhantomCiphertext& encrypted1,
                       const PhantomCiphertext& encrypted2) {
    if (encrypted1.is_ntt_form() || encrypted2.is_ntt_form()) {
        throw std::invalid_argument("encrypted1 or encrypted2 cannot be in NTT form");
    }

    // Extract encryption parameters.
    const auto& context_data = context.get_context_data(encrypted1.chain_index());
    const auto& parms = context_data.parms();
    const auto& rns_tool = context.get_context_data(encrypted1.chain_index()).gpu_rns_tool();

    const size_t poly_degree = parms.poly_modulus_degree();
    const size_t encrypted1_size = encrypted1.size();
    const size_t encrypted2_size = encrypted2.size();
    const size_t base_q_size = rns_tool.base_Ql().size();
    const size_t base_Bsk_size = rns_tool.base_Bsk().size();
    const size_t dest_size = encrypted1_size + encrypted2_size - 1;

    const DModulus* base_rns = context.gpu_rns_tables().modulus();
    const DModulus* base_Bsk = rns_tool.base_Bsk().base();

    // malloc memory, which needs to be freed at the end of the function.
    Pointer<uint64_t> encrypted1_q, encrypted1_Bsk;
    Pointer<uint64_t> encrypted2_q, encrypted2_Bsk;

    encrypted1_q.acquire(allocate<uint64_t>(global_pool(), dest_size * poly_degree * base_q_size));
    encrypted1_Bsk.acquire(allocate<uint64_t>(global_pool(), dest_size * poly_degree * base_Bsk_size));
    encrypted2_q.acquire(allocate<uint64_t>(global_pool(), encrypted2_size * poly_degree * base_q_size));
    encrypted2_Bsk.acquire(allocate<uint64_t>(global_pool(), encrypted2_size * poly_degree * base_Bsk_size));

    // BEHZ, step 1-3
    BEHZ_mul_1(context, encrypted1, encrypted1_q.get(), encrypted1_Bsk.get());
    if (dest_size != 3 || &encrypted1 != &encrypted2)
        BEHZ_mul_1(context, encrypted2, encrypted2_q.get(), encrypted2_Bsk.get());

    uint64_t gridDimGlb;
    // BEHZ, step 4 Compute the ciphertext polynomial product using dyadic multiplication
    // (c0, c1, c2, ...) * (c0', c1', c2', ...)
    //    = (c0 * c0', c0*c1' + c1*c0', c0*c2'+c1*c1'+c2*c0', ...)
    if (dest_size == 3) {
        gridDimGlb = poly_degree * base_q_size / blockDimGlb.x;
        if (&encrypted1 == &encrypted2)
            tensor_square_2x2_rns_poly<<<gridDimGlb, blockDimGlb>>>(encrypted1_q.get(), base_rns, encrypted1_q.get(),
                                                                    poly_degree, base_q_size);

        else
            tensor_prod_2x2_rns_poly<<<gridDimGlb, blockDimGlb>>>(encrypted1_q.get(), encrypted2_q.get(), base_rns,
                                                                  encrypted1_q.get(), poly_degree, base_q_size);

        gridDimGlb = poly_degree * base_Bsk_size / blockDimGlb.x;
        if (&encrypted1 == &encrypted2)
            tensor_square_2x2_rns_poly<<<gridDimGlb, blockDimGlb>>>(encrypted1_Bsk.get(), base_Bsk,
                                                                    encrypted1_Bsk.get(), poly_degree, base_Bsk_size);
        else
            tensor_prod_2x2_rns_poly<<<gridDimGlb, blockDimGlb>>>(encrypted1_Bsk.get(), encrypted2_Bsk.get(), base_Bsk,
                                                                  encrypted1_Bsk.get(), poly_degree, base_Bsk_size);
    } else {
        gridDimGlb = poly_degree * base_q_size / blockDimGlb.x;
        tensor_prod_mxn_rns_poly<<<gridDimGlb, blockDimGlb>>>(encrypted1_q.get(), encrypted1_size, encrypted2_q.get(),
                                                              encrypted2_size, base_rns, encrypted1_q.get(), dest_size,
                                                              poly_degree, base_q_size);

        gridDimGlb = poly_degree * base_Bsk_size / blockDimGlb.x;
        tensor_prod_mxn_rns_poly<<<gridDimGlb, blockDimGlb>>>(
            encrypted1_Bsk.get(), encrypted1_size, encrypted2_Bsk.get(), encrypted2_size, base_Bsk,
            encrypted1_Bsk.get(), dest_size, poly_degree, base_Bsk_size);
    }

    // BEHZ, step 5: NTT backward
    // Step (6): multiply base q components by t (plain_modulus)
    for (size_t i = 0; i < dest_size; i++) {
        nwt_2d_radix8_backward_inplace_scale(encrypted1_q.get() + i * poly_degree * base_q_size,
                                             context.gpu_rns_tables(), base_q_size, 0, context.plain_modulus(),
                                             context.plain_modulus_shoup());
    }
    for (size_t i = 0; i < dest_size; i++) {
        nwt_2d_radix8_backward_inplace_include_temp_mod_scale(
            encrypted1_Bsk.get() + i * poly_degree * base_Bsk_size, rns_tool.gpu_Bsk_tables(), base_Bsk_size, 0,
            rns_tool.gpu_Bsk_tables().size(), rns_tool.tModBsk(), rns_tool.tModBsk_shoup());
    }

    // Resize encrypted1 to destination size
    encrypted1.resize(context, encrypted1.chain_index(), dest_size);

    Pointer<uint64_t> temp;
    temp.acquire(allocate<uint64_t>(global_pool(), poly_degree * base_Bsk_size));
    for (size_t i = 0; i < dest_size; i++) {
        uint64_t* encrypted1_q_iter = encrypted1_q.get() + i * base_q_size * poly_degree;
        uint64_t* encrypted1_Bsk_iter = encrypted1_Bsk.get() + i * base_Bsk_size * poly_degree;
        uint64_t* encrypted1_iter = encrypted1.data() + i * base_q_size * poly_degree;
        // Step (7): divide by q and floor, producing a result(stored in encrypted2_Bsk) in base Bsk
        rns_tool.fast_floor(encrypted1_q_iter, encrypted1_Bsk_iter, temp.get());
        // encrypted1_q is used to avoid malloc in fastbconv_sk
        // Step (8): use Shenoy-Kumaresan method to convert the result to base q and write to encrypted1
        rns_tool.fastbconv_sk(temp.get(), encrypted1_iter);
        // encrypted1_q is used to avoid malloc in fastbconv_sk
    }
    temp.release();
}

size_t FindLevelsToDrop(const PhantomContext& context, size_t multiplicativeDepth, double dcrtBits, bool isKeySwitch,
                        bool isAsymmetric) {
    // Extract encryption parameters.
    auto& context_data = context.get_context_data(0);
    auto& parms = context_data.parms();
    auto n = parms.poly_modulus_degree();

    // handle no relin scenario
    size_t gpu_rns_tool_index = 0;
    if (context.using_keyswitching() == true) {
        gpu_rns_tool_index = 1;
    }

    auto& rns_tool = context.get_context_data(gpu_rns_tool_index).gpu_rns_tool(); // BFV does not drop modulus
    auto mul_tech = rns_tool.mul_tech();

    if (mul_tech != mul_tech_type::hps_overq_leveled)
        throw invalid_argument("FindLevelsToDrop is only used in HPS over Q Leveled");

    double sigma = distributionParameter;
    double alpha = assuranceMeasure;

    double p = parms.plain_modulus().value();

    uint32_t k = rns_tool.size_P();
    uint32_t numPartQ = rns_tool.v_base_part_Ql_to_compl_part_QlP_conv().size();
    uint32_t thresholdParties = 1;
    // Bkey set to thresholdParties * 1 for ternary distribution
    const double Bkey = thresholdParties;

    double w = pow(2, dcrtBits);

    // Bound of the Gaussian error polynomial
    double Berr = sigma * sqrt(alpha);

    // expansion factor delta
    auto delta = [](uint32_t n) -> double { return (2. * sqrt(n)); };

    // norm of fresh ciphertext polynomial (for EXTENDED the noise is reduced to modulus switching noise)
    auto Vnorm = [&](uint32_t n) -> double {
        if (isAsymmetric)
            return (1. + delta(n) * Bkey) / 2.;
        return Berr * (1. + 2. * delta(n) * Bkey);
    };

    auto noiseKS = [&](uint32_t n, double logqPrev, double w) -> double {
        return k * (numPartQ * delta(n) * Berr + delta(n) * Bkey + 1.0) / 2;
    };

    // function used in the EvalMult constraint
    auto C1 = [&](uint32_t n) -> double { return delta(n) * delta(n) * p * Bkey; };

    // function used in the EvalMult constraint
    auto C2 = [&](uint32_t n, double logqPrev) -> double {
        return delta(n) * delta(n) * Bkey * Bkey / 2.0 + noiseKS(n, logqPrev, w);
    };

    // main correctness constraint
    auto logqBFV = [&](uint32_t n, double logqPrev) -> double {
        if (multiplicativeDepth > 0) {
            return log(4 * p) + (multiplicativeDepth - 1) * log(C1(n)) +
                   log(C1(n) * Vnorm(n) + multiplicativeDepth * C2(n, logqPrev));
        }
        return log(p * (4 * (Vnorm(n))));
    };

    // initial values
    double logqPrev = 6. * log(10);
    double logq = logqBFV(n, logqPrev);

    while (fabs(logq - logqPrev) > log(1.001)) {
        logqPrev = logq;
        logq = logqBFV(n, logqPrev);
    }

    // get an estimate of the error q / (4t)
    double loge = logq / log(2) - 2 - log2(p);

    double logExtra = isKeySwitch ? log2(noiseKS(n, logq, w)) : log2(delta(n));

    // adding the cushon to the error (see Appendix D of https://eprint.iacr.org/2021/204.pdf for details)
    // adjusted empirical parameter to 16 from 4 for threshold scenarios to work correctly, this might need to
    // be further refined
    int32_t levels = std::floor((loge - 2 * multiplicativeDepth - 16 - logExtra) / dcrtBits);
    auto sizeQ = static_cast<int32_t>(rns_tool.base_Q().size());

    if (levels < 0)
        levels = 0;
    else if (levels > sizeQ - 1)
        levels = sizeQ - 1;

    return levels;
}

// encrypted1 = encrypted1 * encrypted2
// (c0, c1) * (c0', c1') = (c0*c0', c0'c1+c0c1', c1c1')
// HPS
void bfv_multiply_hps(const PhantomContext& context, PhantomCiphertext& encrypted1,
                      const PhantomCiphertext& encrypted2) {
    if (encrypted1.is_ntt_form() || encrypted2.is_ntt_form()) {
        throw std::invalid_argument("encrypted1 or encrypted2 cannot be in NTT form");
    }

    // Extract encryption parameters.
    const auto& context_data = context.get_context_data(encrypted1.chain_index());
    const auto& parms = context_data.parms();
    const auto n = parms.poly_modulus_degree();
    const auto mul_tech = parms.mul_tech();

    const size_t ct1_size = encrypted1.size_;
    const size_t ct2_size = encrypted2.size_;
    const size_t dest_size = ct1_size + ct2_size - 1;
    if (dest_size != 3)
        throw std::logic_error("dest_size must be 3 when computing BFV multiplication using HPS");

    // Resize encrypted1 to destination size
    encrypted1.resize(context, encrypted1.chain_index(), dest_size);

    // HPS and HPSOverQ does not drop modulus
    uint32_t levelsDropped = 0;

    // handle no relin scenario
    size_t gpu_rns_tool_index = 0;
    if (context.using_keyswitching() == true) {
        gpu_rns_tool_index = 1;
    }

    if (mul_tech == mul_tech_type::hps_overq_leveled) {
        const size_t c1depth = encrypted1.GetNoiseScaleDeg();
        const size_t c2depth = encrypted2.GetNoiseScaleDeg();

        const bool is_Asymmetric = encrypted1.is_asymmetric();
        const size_t levels = std::max(c1depth, c2depth) - 1;
        const auto dcrtBits = static_cast<double>(context.get_context_data(gpu_rns_tool_index).gpu_rns_tool().qMSB());

        // how many levels to drop
        levelsDropped = FindLevelsToDrop(context, levels, dcrtBits, false, is_Asymmetric);
    }

    const auto& rns_tool = context.get_context_data(gpu_rns_tool_index + levelsDropped).gpu_rns_tool();
    const DModulus* base_QlRl = rns_tool.base_QlRl().base();
    const auto& gpu_QlRl_tables = rns_tool.gpu_QlRl_tables();
    const size_t size_Q = rns_tool.base_Q().size();
    const size_t size_Ql = rns_tool.base_Ql().size();
    const size_t size_Rl = rns_tool.base_Rl().size();
    const size_t size_QlRl = size_Ql + size_Rl;

    /* --------------------------------- ct1 BConv -------------------------------- */
    Pointer<uint64_t> ct1;
    ct1.acquire(allocate<uint64_t>(global_pool(), dest_size * size_QlRl * n));
    for (size_t i = 0; i < ct1_size; i++) {
        const uint64_t* encrypted1_ptr = encrypted1.data() + i * size_Q * n;
        uint64_t* ct1_ptr = ct1.get() + i * size_QlRl * n;
        uint64_t* ct1_Ql_ptr = ct1_ptr;
        uint64_t* ct1_Rl_ptr = ct1_Ql_ptr + size_Ql * n;

        if (mul_tech == mul_tech_type::hps_overq_leveled && levelsDropped)
            rns_tool.scaleAndRound_HPS_Q_Ql(ct1_Ql_ptr, encrypted1_ptr);
        else
            CUDA_CHECK(
            hipMemcpy(ct1_Ql_ptr, encrypted1_ptr, size_Ql * n * sizeof(uint64_t), hipMemcpyDeviceToDevice));

        rns_tool.base_Ql_to_Rl_conv().bConv_HPS(ct1_Rl_ptr, ct1_Ql_ptr, n);
    }

    if (&encrypted1 == &encrypted2) {
        // if square, no need to compute ct2
        /* --------------------------------- ct1 *= ct1 -------------------------------- */
        // forward NTT
        for (size_t i = 0; i < ct1_size; i++) {
            uint64_t* ct1_ptr = ct1.get() + i * size_QlRl * n;
            nwt_2d_radix8_forward_inplace(ct1_ptr, gpu_QlRl_tables, size_QlRl, 0);
        }

        // (c0, c1, c2, ...) * (c0', c1', c2', ...)
        //    = (c0 * c0', c0*c1' + c1*c0', c0*c2'+c1*c1'+c2*c0', ...)
        uint64_t gridDimGlb = n * size_QlRl / blockDimGlb.x;
        tensor_square_2x2_rns_poly<<<gridDimGlb, blockDimGlb>>>(ct1.get(), base_QlRl, ct1.get(), n, size_QlRl);
    } else {
        /* --------------------------------- ct2 BConv -------------------------------- */
        Pointer<uint64_t> ct2;
        // allocate enough space
        ct2.acquire(allocate<uint64_t>(global_pool(), ct2_size * size_QlRl * n));
        for (size_t i = 0; i < ct2_size; i++) {
            const uint64_t* encrypted2_ptr = encrypted2.data() + i * size_Q * n;
            uint64_t* ct2_ptr = ct2.get() + i * size_QlRl * n;
            uint64_t* ct2_Ql_ptr = ct2_ptr;
            uint64_t* ct2_Rl_ptr = ct2_Ql_ptr + size_Ql * n;

            if (mul_tech == mul_tech_type::hps) {
                CUDA_CHECK(hipMemcpy(ct2_Ql_ptr, encrypted2_ptr, size_Ql * n * sizeof(uint64_t),
                    hipMemcpyDeviceToDevice));
                rns_tool.base_Ql_to_Rl_conv().bConv_HPS(ct2_Rl_ptr, ct2_Ql_ptr, n);
            } else if (mul_tech == mul_tech_type::hps_overq || mul_tech == mul_tech_type::hps_overq_leveled) {
                if (levelsDropped)
                    rns_tool.base_Q_to_Rl_conv().bConv_BEHZ_var1(ct2_Rl_ptr, encrypted2_ptr, n);
                else
                    rns_tool.base_Ql_to_Rl_conv().bConv_BEHZ_var1(ct2_Rl_ptr, encrypted2_ptr, n);
                rns_tool.base_Rl_to_Ql_conv().bConv_HPS(ct2_Ql_ptr, ct2_Rl_ptr, n);
            }
        }

        /* --------------------------------- ct1 *= ct2 -------------------------------- */
        // forward NTT
        for (size_t i = 0; i < ct1_size; i++) {
            uint64_t* ct1_ptr = ct1.get() + i * size_QlRl * n;
            nwt_2d_radix8_forward_inplace(ct1_ptr, gpu_QlRl_tables, size_QlRl, 0);
        }

        for (size_t i = 0; i < ct2_size; i++) {
            uint64_t* ct2_ptr = ct2.get() + i * size_QlRl * n;
            nwt_2d_radix8_forward_inplace(ct2_ptr, gpu_QlRl_tables, size_QlRl, 0);
        }

        // (c0, c1, c2, ...) * (c0', c1', c2', ...)
        //    = (c0 * c0', c0*c1' + c1*c0', c0*c2'+c1*c1'+c2*c0', ...)
        uint64_t gridDimGlb = n * size_QlRl / blockDimGlb.x;
        tensor_prod_2x2_rns_poly<<<gridDimGlb, blockDimGlb>>>(ct1.get(), ct2.get(), base_QlRl, ct1.get(), n, size_QlRl);
    }

    // inverse NTT
    for (size_t i = 0; i < dest_size; i++) {
        uint64_t* ct1_ptr = ct1.get() + i * size_QlRl * n;
        nwt_2d_radix8_backward_inplace(ct1_ptr, gpu_QlRl_tables, size_QlRl, 0);
    }

    /* --------------------------------- ct1 BConv -------------------------------- */
    // scale and round
    for (size_t i = 0; i < dest_size; i++) {
        uint64_t* encrypted1_ptr = encrypted1.data() + i * size_Q * n;
        const uint64_t* ct1_ptr = ct1.get() + i * size_QlRl * n;
        if (mul_tech == mul_tech_type::hps) {
            Pointer<uint64_t> temp;
            temp.acquire(allocate<uint64_t>(global_pool(), size_Rl * n));
            // scale and round QlRl to Rl
            rns_tool.scaleAndRound_HPS_QR_R(temp.get(), ct1_ptr);
            // Rl -> Ql
            rns_tool.base_Rl_to_Ql_conv().bConv_HPS(encrypted1_ptr, temp.get(), n);
            temp.release();
        } else if (mul_tech == mul_tech_type::hps_overq || mul_tech == mul_tech_type::hps_overq_leveled) {
            // scale and round QlRl to Ql
            rns_tool.scaleAndRound_HPS_QlRl_Ql(encrypted1_ptr, ct1_ptr);

            if (levelsDropped) {
                rns_tool.ExpandCRTBasis_Ql_Q(encrypted1_ptr, encrypted1_ptr);
            }
        }
    }

    if (mul_tech == mul_tech_type::hps_overq_leveled) {
        encrypted1.SetNoiseScaleDeg(std::max(encrypted1.GetNoiseScaleDeg(), encrypted2.GetNoiseScaleDeg()) + 1);
    }
}

// encrypted1 = encrypted1 * encrypted2
// (c0, c1) * (c0', c1') = (c0*c0', c0'c1+c0c1', c1c1')
void bfv_multiply(const PhantomContext& context, PhantomCiphertext& encrypted1, const PhantomCiphertext& encrypted2) {
    auto mul_tech = context.mul_tech();
    if (mul_tech == mul_tech_type::behz) {
        bfv_multiply_behz(context, encrypted1, encrypted2);
    } else if (mul_tech == mul_tech_type::hps || mul_tech == mul_tech_type::hps_overq ||
               mul_tech == mul_tech_type::hps_overq_leveled) {
        bfv_multiply_hps(context, encrypted1, encrypted2);
    } else {
        throw invalid_argument("mul_tech not supported for bfv_multiply");
    }
}

void bfv_mul_relin_hps(const PhantomContext& context, PhantomCiphertext& encrypted1,
                       const PhantomCiphertext& encrypted2, const PhantomRelinKey& relin_keys) {
    if (encrypted1.is_ntt_form() || encrypted2.is_ntt_form()) {
        throw std::invalid_argument("encrypted1 or encrypted2 cannot be in NTT form");
    }

    // Extract encryption parameters.
    const auto& context_data = context.get_context_data(encrypted1.chain_index());
    const auto& parms = context_data.parms();
    const auto n = parms.poly_modulus_degree();
    const auto mul_tech = parms.mul_tech();

    const size_t ct1_size = encrypted1.size_;
    const size_t ct2_size = encrypted2.size_;
    const size_t dest_size = ct1_size + ct2_size - 1;
    if (dest_size != 3)
        throw std::logic_error("dest_size must be 3 when computing BFV multiplication using HPS");

    // Resize encrypted1 to destination size
    encrypted1.resize(context, encrypted1.chain_index(), dest_size);

    // HPS and HPSOverQ does not drop modulus
    uint32_t levelsDropped = 0;

    if (mul_tech == mul_tech_type::hps_overq_leveled) {
        const size_t c1depth = encrypted1.GetNoiseScaleDeg();
        const size_t c2depth = encrypted2.GetNoiseScaleDeg();

        const bool is_Asymmetric = encrypted1.is_asymmetric();
        const size_t levels = std::max(c1depth, c2depth) - 1;
        const auto dcrtBits = static_cast<double>(context.get_context_data(1).gpu_rns_tool().qMSB());

        // how many levels to drop
        levelsDropped = FindLevelsToDrop(context, levels, dcrtBits, false, is_Asymmetric);
    }

    const auto& rns_tool = context.get_context_data(1 + levelsDropped).gpu_rns_tool();
    const DModulus* base_QlRl = rns_tool.base_QlRl().base();
    const auto& gpu_QlRl_tables = rns_tool.gpu_QlRl_tables();
    const size_t size_Q = rns_tool.base_Q().size();
    const size_t size_Ql = rns_tool.base_Ql().size();
    const size_t size_Rl = rns_tool.base_Rl().size();
    const size_t size_QlRl = size_Ql + size_Rl;

    /* --------------------------------- ct1 BConv -------------------------------- */
    Pointer<uint64_t> ct1;
    ct1.acquire(allocate<uint64_t>(global_pool(), dest_size * size_QlRl * n));
    for (size_t i = 0; i < ct1_size; i++) {
        const uint64_t* encrypted1_ptr = encrypted1.data() + i * size_Q * n;
        uint64_t* ct1_ptr = ct1.get() + i * size_QlRl * n;
        uint64_t* ct1_Ql_ptr = ct1_ptr;
        uint64_t* ct1_Rl_ptr = ct1_Ql_ptr + size_Ql * n;

        if (mul_tech == mul_tech_type::hps_overq_leveled && levelsDropped)
            rns_tool.scaleAndRound_HPS_Q_Ql(ct1_Ql_ptr, encrypted1_ptr);
        else
            CUDA_CHECK(
            hipMemcpy(ct1_Ql_ptr, encrypted1_ptr, size_Ql * n * sizeof(uint64_t), hipMemcpyDeviceToDevice));

        rns_tool.base_Ql_to_Rl_conv().bConv_HPS(ct1_Rl_ptr, ct1_Ql_ptr, n);
    }

    if (&encrypted1 == &encrypted2) {
        // square
        /* --------------------------------- ct1 *= ct1 -------------------------------- */
        // forward NTT
        for (size_t i = 0; i < ct1_size; i++) {
            uint64_t* ct1_ptr = ct1.get() + i * size_QlRl * n;
            nwt_2d_radix8_forward_inplace(ct1_ptr, gpu_QlRl_tables, size_QlRl, 0);
        }

        // (c0, c1, c2, ...) * (c0', c1', c2', ...)
        //    = (c0 * c0', c0*c1' + c1*c0', c0*c2'+c1*c1'+c2*c0', ...)
        uint64_t gridDimGlb = n * size_QlRl / blockDimGlb.x;
        tensor_square_2x2_rns_poly<<<gridDimGlb, blockDimGlb>>>(ct1.get(), base_QlRl, ct1.get(), n, size_QlRl);
    } else {
        /* --------------------------------- ct2 BConv -------------------------------- */
        Pointer<uint64_t> ct2;
        // allocate enough space
        ct2.acquire(allocate<uint64_t>(global_pool(), ct2_size * size_QlRl * n));
        for (size_t i = 0; i < ct2_size; i++) {
            const uint64_t* encrypted2_ptr = encrypted2.data() + i * size_Q * n;
            uint64_t* ct2_ptr = ct2.get() + i * size_QlRl * n;
            uint64_t* ct2_Ql_ptr = ct2_ptr;
            uint64_t* ct2_Rl_ptr = ct2_Ql_ptr + size_Ql * n;

            if (mul_tech == mul_tech_type::hps) {
                CUDA_CHECK(hipMemcpy(ct2_Ql_ptr, encrypted2_ptr, size_Ql * n * sizeof(uint64_t),
                    hipMemcpyDeviceToDevice));
                rns_tool.base_Ql_to_Rl_conv().bConv_HPS(ct2_Rl_ptr, ct2_Ql_ptr, n);
            } else if (mul_tech == mul_tech_type::hps_overq || mul_tech == mul_tech_type::hps_overq_leveled) {
                if (levelsDropped)
                    rns_tool.base_Q_to_Rl_conv().bConv_BEHZ_var1(ct2_Rl_ptr, encrypted2_ptr, n);
                else
                    rns_tool.base_Ql_to_Rl_conv().bConv_BEHZ_var1(ct2_Rl_ptr, encrypted2_ptr, n);
                rns_tool.base_Rl_to_Ql_conv().bConv_HPS(ct2_Ql_ptr, ct2_Rl_ptr, n);
            }
        }

        /* --------------------------------- ct1 *= ct2 -------------------------------- */
        // forward NTT
        for (size_t i = 0; i < ct1_size; i++) {
            uint64_t* ct1_ptr = ct1.get() + i * size_QlRl * n;
            nwt_2d_radix8_forward_inplace(ct1_ptr, gpu_QlRl_tables, size_QlRl, 0);
        }

        for (size_t i = 0; i < ct2_size; i++) {
            uint64_t* ct2_ptr = ct2.get() + i * size_QlRl * n;
            nwt_2d_radix8_forward_inplace(ct2_ptr, gpu_QlRl_tables, size_QlRl, 0);
        }

        // (c0, c1, c2, ...) * (c0', c1', c2', ...)
        //    = (c0 * c0', c0*c1' + c1*c0', c0*c2'+c1*c1'+c2*c0', ...)
        uint64_t gridDimGlb = n * size_QlRl / blockDimGlb.x;
        tensor_prod_2x2_rns_poly<<<gridDimGlb, blockDimGlb>>>(ct1.get(), ct2.get(), base_QlRl, ct1.get(), n, size_QlRl);
    }

    // inverse NTT
    for (size_t i = 0; i < dest_size; i++) {
        uint64_t* ct1_ptr = ct1.get() + i * size_QlRl * n;
        nwt_2d_radix8_backward_inplace(ct1_ptr, gpu_QlRl_tables, size_QlRl, 0);
    }

    /* --------------------------------- ct1 BConv -------------------------------- */
    // scale and round
    for (size_t i = 0; i < dest_size; i++) {
        uint64_t* encrypted1_ptr = encrypted1.data() + i * size_Q * n;
        const uint64_t* ct1_ptr = ct1.get() + i * size_QlRl * n;
        if (mul_tech == mul_tech_type::hps) {
            Pointer<uint64_t> temp;
            temp.acquire(allocate<uint64_t>(global_pool(), size_Rl * n));
            // scale and round QlRl to Rl
            rns_tool.scaleAndRound_HPS_QR_R(temp.get(), ct1_ptr);
            // Rl -> Ql
            rns_tool.base_Rl_to_Ql_conv().bConv_HPS(encrypted1_ptr, temp.get(), n);
        } else if (mul_tech == mul_tech_type::hps_overq || mul_tech == mul_tech_type::hps_overq_leveled) {
            // scale and round QlRl to Ql
            rns_tool.scaleAndRound_HPS_QlRl_Ql(encrypted1_ptr, ct1_ptr);

            if (levelsDropped && i != dest_size - 1) {
                rns_tool.ExpandCRTBasis_Ql_Q(encrypted1_ptr, encrypted1_ptr);
            }
        }
    }

    if (mul_tech == mul_tech_type::hps_overq_leveled) {
        encrypted1.SetNoiseScaleDeg(std::max(encrypted1.GetNoiseScaleDeg(), encrypted2.GetNoiseScaleDeg()) + 1);
    }

    // Extract encryption parameters.
    const size_t decomp_modulus_size = parms.coeff_modulus().size();
    const auto& key_vector = relin_keys.public_keys_;
    const auto key_component_count = key_vector[0].pk_.size_;
    const auto scheme = parms.scheme();

    // Verify parameters.
    if (encrypted1.size() != 3) {
        throw invalid_argument("destination_size must be 3");
    }
    if (scheme == scheme_type::bfv && encrypted1.is_ntt_form_) {
        throw invalid_argument("BFV encrypted cannot be in NTT form");
    }
    if (key_component_count != 2) {
        throw invalid_argument("destination_size must be equal to key_component_count");
    }

    // only c2 is not scale&round to Ql
    const uint64_t* c2 = encrypted1.data() + 2 * size_Q * n;

    // Extract encryption parameters.
    const auto& key_context_data = context.get_context_data(0);
    const auto& key_parms = key_context_data.parms();
    const auto& key_modulus = key_parms.coeff_modulus();
    const auto& modulus_QP = context.gpu_rns_tables().modulus();
    const size_t size_P = key_parms.special_modulus_size();
    const size_t size_QP = key_modulus.size();
    const size_t size_QlP = size_Ql + size_P;
    const size_t size_Ql_n = size_Ql * n;
    const size_t size_QP_n = size_QP * n;
    const size_t size_QlP_n = size_QlP * n;
    const size_t beta = rns_tool.v_base_part_Ql_to_compl_part_QlP_conv().size();

    // mod up
    Pointer<uint64_t> t_mod_up;
    t_mod_up.acquire(allocate<uint64_t>(global_pool(), beta * size_QlP_n));
    rns_tool.modup(t_mod_up.get(), c2, context.gpu_rns_tables(), scheme);

    // key switch
    Pointer<uint64_t> cx;
    cx.acquire(allocate<uint64_t>(global_pool(), 2 * size_QlP_n));
    auto reduction_threshold = (1 << (bits_per_uint64 - rns_tool.qMSB() - 1)) - 1;
    key_switch_inner_prod_c2_and_evk<<<size_QlP_n / blockDimGlb.x, blockDimGlb>>>(
        cx.get(), t_mod_up.get(), relin_keys.public_keys_ptr_.get(), modulus_QP, n, size_QP, size_QP_n, size_QlP,
        size_QlP_n, size_Q, size_Ql, beta, reduction_threshold);

    // mod down
    for (size_t i = 0; i < 2; i++) {
        const auto cx_i = cx.get() + i * size_QlP_n;
        rns_tool.moddown_from_NTT(cx_i, cx_i, context.gpu_rns_tables(), scheme);
    }

    for (size_t i = 0; i < 2; i++) {
        const auto cx_i = cx.get() + i * size_QlP_n;

        if (mul_tech == mul_tech_type::hps_overq_leveled && levelsDropped) {
            auto ct_i = encrypted1.data() + i * size_Q * n;
            rns_tool.ExpandCRTBasis_Ql_Q_add_to_ct(ct_i, cx_i);
        } else {
            auto ct_i = encrypted1.data() + i * size_Ql_n;
            add_to_ct_kernel<<<size_Ql_n / blockDimGlb.x, blockDimGlb>>>(ct_i, cx_i, rns_tool.base_Ql().base(), n,
                                                                         size_Ql);
        }
    }

    // update the encrypted
    encrypted1.resize(key_component_count, decomp_modulus_size, n);
}

// encrypted1 = encrypted1 * encrypted2
void multiply_inplace(const PhantomContext& context, PhantomCiphertext& encrypted1,
                      const PhantomCiphertext& encrypted2) {
    // Verify parameters.
    if (encrypted1.parms_id() != encrypted2.parms_id()) {
        throw invalid_argument("encrypted1 and encrypted2 parameter mismatch");
    }
    if (encrypted1.chain_index() != encrypted2.chain_index())
        throw std::invalid_argument("encrypted1 and encrypted2 parameter mismatch");
    if (encrypted1.is_ntt_form() != encrypted2.is_ntt_form())
        throw std::invalid_argument("NTT form mismatch");
    // if (encrypted1.scale() != encrypted2.scale())
    //     throw std::invalid_argument("scale mismatch");
    if (encrypted1.size_ != encrypted2.size_)
        throw std::invalid_argument("poly number mismatch");

    // Extract encryption parameters.
    auto& context_data = context.get_context_data(encrypted1.chain_index());
    auto& parms = context_data.parms();

    switch (parms.scheme()) {
        case scheme_type::bfv:
            bfv_multiply(context, encrypted1, encrypted2);
            break;
        case scheme_type::ckks:
        case scheme_type::bgv:
            bgv_ckks_multiply(context, encrypted1, encrypted2);
            break;

        default:
            throw invalid_argument("unsupported scheme");
    }
}

// encrypted1 = encrypted1 * encrypted2
// relin(encrypted1)
void multiply_and_relin_inplace(const PhantomContext& context, PhantomCiphertext& encrypted1,
                                const PhantomCiphertext& encrypted2, const PhantomRelinKey& relin_keys) {
    // Verify parameters.
    if (encrypted1.parms_id() != encrypted2.parms_id()) {
        throw invalid_argument("encrypted1 and encrypted2 parameter mismatch");
    }
    if (encrypted1.chain_index() != encrypted2.chain_index())
        throw std::invalid_argument("encrypted1 and encrypted2 parameter mismatch");
    if (encrypted1.is_ntt_form() != encrypted2.is_ntt_form())
        throw std::invalid_argument("NTT form mismatch");
    if (encrypted1.scale() != encrypted2.scale())
        throw std::invalid_argument("scale mismatch");
    if (encrypted1.size_ != encrypted2.size_)
        throw std::invalid_argument("poly number mismatch");

    // Extract encryption parameters.
    auto& context_data = context.get_context_data(encrypted1.chain_index());
    auto& parms = context_data.parms();
    auto scheme = parms.scheme();
    auto mul_tech = parms.mul_tech();

    switch (scheme) {
        case scheme_type::bfv:
            if (mul_tech == mul_tech_type::hps || mul_tech == mul_tech_type::hps_overq ||
                mul_tech == mul_tech_type::hps_overq_leveled) {
                // enable fast mul&relin
                bfv_mul_relin_hps(context, encrypted1, encrypted2, relin_keys);
            } else if (mul_tech == mul_tech_type::behz) {
                bfv_multiply_behz(context, encrypted1, encrypted2);
                relinearize_inplace(context, encrypted1, relin_keys);
            } else {
                throw invalid_argument("unsupported mul tech in BFV mul&relin");
            }
            break;

        case scheme_type::ckks:
        case scheme_type::bgv:
            bgv_ckks_multiply(context, encrypted1, encrypted2);
            relinearize_inplace(context, encrypted1, relin_keys);
            break;

        default:
            throw invalid_argument("unsupported scheme");
    }
}

void add_plain_inplace(const PhantomContext& context, PhantomCiphertext& encrypted, const PhantomPlaintext& plain) {
    // Extract encryption parameters.
    auto& context_data = context.get_context_data(encrypted.chain_index());
    auto& parms = context_data.parms();

    if (parms.scheme() == scheme_type::bfv && encrypted.is_ntt_form()) {
        throw std::invalid_argument("BFV encrypted cannot be in NTT form");
    }
    if (parms.scheme() == scheme_type::ckks && !(encrypted.is_ntt_form())) {
        throw std::invalid_argument("CKKS encrypted must be in NTT form");
    }
    if (parms.scheme() == scheme_type::bgv && !(encrypted.is_ntt_form())) {
        throw std::invalid_argument("BGV encrypted must be in NTT form");
    }
    if (encrypted.scale() != plain.scale()) {
        // TODO: be more precious
        throw std::invalid_argument("scale mismatch");
    }

    auto& coeff_modulus = parms.coeff_modulus();
    auto coeff_mod_size = coeff_modulus.size();
    auto poly_degree = parms.poly_modulus_degree();
    auto base_rns = context.gpu_rns_tables().modulus();

    uint64_t gridDimGlb = poly_degree * coeff_mod_size / blockDimGlb.x;

    switch (parms.scheme()) {
        case scheme_type::bfv: {
            multiply_add_plain_with_scaling_variant(context, plain, encrypted.chain_index(), encrypted);
            break;
        }
        case scheme_type::ckks: {
            // (c0 + pt, c1)
            add_rns_poly<<<gridDimGlb, blockDimGlb>>>(encrypted.data(), plain.data(), base_rns, encrypted.data(),
                                                      poly_degree, coeff_mod_size);
            break;
        }
        case scheme_type::bgv: {
            // TODO: make bgv plaintext is_ntt_form true?
            // c0 = c0 + plaintext
            Pointer<uint64_t> plain_copy;
            plain_copy.acquire(allocate<uint64_t>(global_pool(), coeff_mod_size * poly_degree));
            for (size_t i = 0; i < coeff_mod_size; i++) {
                // modup t -> {q0, q1, ...., qj}
                nwt_2d_radix8_forward_modup_fuse(plain_copy.get() + i * poly_degree, plain.data(), i,
                                                 context.gpu_rns_tables(), 1, 0);
            }
            // (c0 + pt, c1)
            multiply_scalar_and_add_rns_poly<<<gridDimGlb, blockDimGlb>>>(
                encrypted.data(), plain_copy.get(), encrypted.correction_factor(), base_rns, encrypted.data(),
                poly_degree, coeff_mod_size);
            break;
        }
        default:
            throw invalid_argument("unsupported scheme");
    }
}

void sub_plain_inplace(const PhantomContext& context, PhantomCiphertext& encrypted, const PhantomPlaintext& plain) {
    // Extract encryption parameters.
    auto& context_data = context.get_context_data(encrypted.chain_index());
    auto& parms = context_data.parms();

    if (parms.scheme() == scheme_type::bfv && encrypted.is_ntt_form()) {
        throw std::invalid_argument("BFV encrypted cannot be in NTT form");
    }
    if (parms.scheme() == scheme_type::ckks && !(encrypted.is_ntt_form())) {
        throw std::invalid_argument("CKKS encrypted must be in NTT form");
    }
    if (parms.scheme() == scheme_type::bgv && !(encrypted.is_ntt_form())) {
        throw std::invalid_argument("BGV encrypted must be in NTT form");
    }
    if (encrypted.scale() != plain.scale()) {
        // TODO: be more precious
        throw std::invalid_argument("scale mismatch");
    }

    auto& coeff_modulus = parms.coeff_modulus();
    auto coeff_mod_size = coeff_modulus.size();
    auto poly_degree = parms.poly_modulus_degree();
    auto base_rns = context.gpu_rns_tables().modulus();
    uint64_t gridDimGlb = poly_degree * coeff_mod_size / blockDimGlb.x;

    switch (parms.scheme()) {
        case scheme_type::bfv: {
            multiply_sub_plain_with_scaling_variant(context, plain, encrypted.chain_index(), encrypted);
            break;
        }
        case scheme_type::ckks: {
            // (c0 - pt, c1)
            sub_rns_poly<<<gridDimGlb, blockDimGlb>>>(encrypted.data(), plain.data(), base_rns, encrypted.data(),
                                                      poly_degree, coeff_mod_size);
            break;
        }
        case scheme_type::bgv: {
            // TODO: make bgv plaintext is_ntt_form true?
            // c0 = c0 - plaintext
            Pointer<uint64_t> plain_copy;
            plain_copy.acquire(allocate<uint64_t>(global_pool(), coeff_mod_size * poly_degree));
            for (size_t i = 0; i < coeff_mod_size; i++) {
                // modup t -> {q0, q1, ...., qj}
                nwt_2d_radix8_forward_modup_fuse(plain_copy.get() + i * poly_degree, plain.data(), i,
                                                 context.gpu_rns_tables(), 1, 0);
            }
            // (c0 - pt, c1)
            multiply_scalar_and_sub_rns_poly<<<gridDimGlb, blockDimGlb>>>(
                encrypted.data(), plain_copy.get(), encrypted.correction_factor(), base_rns, encrypted.data(),
                poly_degree, coeff_mod_size);
            break;
        }
        default:
            throw invalid_argument("unsupported scheme");
    }
}

void multiply_plain_ntt(const PhantomContext& context, PhantomCiphertext& encrypted, const PhantomPlaintext& plain) {
    if (!plain.is_ntt_form()) {
        throw invalid_argument("plain_ntt is not in NTT form");
    }
    if (encrypted.chain_index() != plain.chain_index()) {
        throw std::invalid_argument("encrypted and plain parameter mismatch");
    }
    if (encrypted.parms_id() != plain.parms_id()) {
        throw invalid_argument("encrypted_ntt and plain_ntt parameter mismatch");
    }

    // Extract encryption parameters.
    auto& context_data = context.get_context_data(encrypted.chain_index());
    auto& parms = context_data.parms();
    auto& coeff_modulus = parms.coeff_modulus();
    auto coeff_mod_size = coeff_modulus.size();
    auto poly_degree = parms.poly_modulus_degree();
    auto base_rns = context.gpu_rns_tables().modulus();
    auto rns_coeff_count = poly_degree * coeff_mod_size;

    double new_scale = encrypted.scale() * plain.scale();

    //(c0 * pt, c1 * pt)
    for (size_t i = 0; i < encrypted.size(); i++) {
        uint64_t gridDimGlb = poly_degree * coeff_mod_size / blockDimGlb.x;
        multiply_rns_poly<<<gridDimGlb, blockDimGlb>>>(encrypted.data() + i * rns_coeff_count, plain.data(), base_rns,
                                                       encrypted.data() + i * rns_coeff_count, poly_degree,
                                                       coeff_mod_size);
    }

    encrypted.scale() = new_scale;
}

void multiply_plain_normal(const PhantomContext& context, PhantomCiphertext& encrypted, const PhantomPlaintext& plain) {
    // Extract encryption parameters.
    auto& context_data = context.get_context_data(encrypted.chain_index());
    auto& parms = context_data.parms();
    auto& coeff_modulus = parms.coeff_modulus();
    auto coeff_mod_size = coeff_modulus.size();
    auto poly_degree = parms.poly_modulus_degree();
    auto rns_coeff_count = poly_degree * coeff_mod_size;
    auto base_rns = context.gpu_rns_tables().modulus();
    auto encrypted_size = encrypted.size();

    auto plain_upper_half_threshold = context_data.plain_upper_half_threshold();
    auto plain_upper_half_increment = context.plain_upper_half_increment();

    double new_scale = encrypted.scale() * plain.scale();

    /*
    !!! This optimizations is removed also due to the access of device memory in host
    Optimizations for constant / monomial multiplication can lead to the presence of a timing side-channel,
    as the processing time varies with the plaintext, and therefore leaks plaintext, which may be sensitive.
    */
    /*if (plain_nonzero_coeff_count == 1)
    {
    }*/

    uint64_t gridDimGlb = rns_coeff_count / blockDimGlb.x;
    // Generic case: any plaintext polynomial
    // Allocate temporary space for an entire RNS polynomial
    Pointer<uint64_t> temp;
    temp.acquire(allocate<uint64_t>(global_pool(), rns_coeff_count));

    // if (context_data.qualifiers().using_fast_plain_lift) {
    // if t is smaller than every qi
    abs_plain_rns_poly<<<gridDimGlb, blockDimGlb>>>(plain.data(), plain_upper_half_threshold,
                                                    plain_upper_half_increment, temp.get(), poly_degree,
                                                    coeff_mod_size);
    // }
    // else {
    //     // need to perform decompose
    //     // N-slot plain, for each slot,
    //     // temp["coeff_mod_size"] = plain[tid] + plain_upper_half_increment (i.e., q-t) when (plain[tid] >=
    //     plain_upper_half_threshold)
    //     // otherwise temp["coeff_mod_size"] = plain[tid]
    //     auto &rns_tool = context.get_context_data(encrypted.chain_index()).gpu_rns_tool();
    //     auto &base_q = rns_tool.base_Ql_;
    //     // each block (num is N) of coeff_mod_size size, is decomposed into N * coeff_mod_size data
    //     base_q.decompose_array(temp.get(), plain.data(), base_rns, poly_degree, plain_upper_half_increment,
    //                            plain_upper_half_threshold);
    // }

    nwt_2d_radix8_forward_inplace(temp.get(), context.gpu_rns_tables(), coeff_mod_size, 0);

    // (c0 * pt, c1 * pt)
    for (size_t i = 0; i < encrypted_size; i++) {
        uint64_t* ci = encrypted.data() + i * rns_coeff_count;
        // NTT
        nwt_2d_radix8_forward_inplace(ci, context.gpu_rns_tables(), coeff_mod_size, 0);
        // Pointwise multiplication
        multiply_rns_poly<<<gridDimGlb, blockDimGlb>>>(ci, temp.get(), base_rns, ci, poly_degree, coeff_mod_size);
        // inverse NTT
        nwt_2d_radix8_backward_inplace(ci, context.gpu_rns_tables(), coeff_mod_size, 0);
    }

    encrypted.scale() = new_scale;
}

void multiply_plain_inplace(const PhantomContext& context, PhantomCiphertext& encrypted,
                            const PhantomPlaintext& plain) {
    // Extract encryption parameters.
    auto& context_data = context.get_context_data(encrypted.chain_index());
    auto& parms = context_data.parms();
    auto scheme = parms.scheme();

    if (scheme == scheme_type::bfv) {
        multiply_plain_normal(context, encrypted, plain);
    } else if (scheme == scheme_type::ckks) {
        multiply_plain_ntt(context, encrypted, plain);
    } else if (scheme == scheme_type::bgv) {
        // Extract encryption parameters.
        auto& coeff_modulus = parms.coeff_modulus();
        auto coeff_mod_size = coeff_modulus.size();
        auto poly_degree = parms.poly_modulus_degree();
        auto base_rns = context.gpu_rns_tables().modulus();
        auto rns_coeff_count = poly_degree * coeff_mod_size;

        Pointer<uint64_t> plain_copy;
        plain_copy.acquire(allocate<uint64_t>(global_pool(), coeff_mod_size * poly_degree));
        for (size_t i = 0; i < coeff_mod_size; i++) {
            // modup t -> {q0, q1, ...., qj}
            nwt_2d_radix8_forward_modup_fuse(plain_copy.get() + i * poly_degree, plain.data(), i,
                                             context.gpu_rns_tables(), 1, 0);
        }

        double new_scale = encrypted.scale() * plain.scale();

        //(c0 * pt, c1 * pt)
        for (size_t i = 0; i < encrypted.size(); i++) {
            uint64_t gridDimGlb = poly_degree * coeff_mod_size / blockDimGlb.x;
            multiply_rns_poly<<<gridDimGlb, blockDimGlb>>>(encrypted.data() + i * rns_coeff_count, plain_copy.get(),
                                                           base_rns, encrypted.data() + i * rns_coeff_count,
                                                           poly_degree, coeff_mod_size);
        }

        encrypted.scale() = new_scale;
    } else {
        throw std::invalid_argument("unsupported scheme");
    }
}

void transform_to_ntt_inplace(const PhantomContext& context, PhantomCiphertext& encrypted) {
    if (encrypted.is_ntt_form()) {
        throw invalid_argument("encrypted is already in NTT form");
    }

    // Extract encryption parameters.
    auto& context_data = context.get_context_data(encrypted.chain_index());
    auto& parms = context_data.parms();
    size_t poly_degree = parms.poly_modulus_degree();
    auto coeff_mod_size = parms.coeff_modulus().size();

    for (size_t i = 0; i < encrypted.size(); i++) {
        // Transform each polynomial to NTT domain
        auto ci = encrypted.data() + i * coeff_mod_size * poly_degree;
        nwt_2d_radix8_forward_inplace(ci, context.gpu_rns_tables(), coeff_mod_size, 0);
    }

    // Finally change the is_ntt_transformed flag
    encrypted.is_ntt_form() = true;
}

void transform_from_ntt_inplace(const PhantomContext& context, PhantomCiphertext& encrypted_ntt) {
    if (!encrypted_ntt.is_ntt_form()) {
        throw invalid_argument("encrypted_ntt is not in NTT form");
    }

    // Extract encryption parameters.
    auto& context_data = context.get_context_data(encrypted_ntt.chain_index());
    auto& parms = context_data.parms();
    size_t poly_degree = parms.poly_modulus_degree();
    auto coeff_mod_size = parms.coeff_modulus().size();

    for (size_t i = 0; i < encrypted_ntt.size(); i++) {
        // Transform each polynomial to NTT domain
        auto ci = encrypted_ntt.data() + i * coeff_mod_size * poly_degree;
        nwt_2d_radix8_backward_inplace(ci, context.gpu_rns_tables(), coeff_mod_size, 0);
    }

    // Finally change the is_ntt_transformed flag
    encrypted_ntt.is_ntt_form() = false;
}

void relinearize_inplace(const PhantomContext& context, PhantomCiphertext& encrypted,
                         const PhantomRelinKey& relin_keys) {
    // Extract encryption parameters.
    auto& context_data = context.get_context_data(encrypted.chain_index());
    auto& parms = context_data.parms();
    size_t decomp_modulus_size = parms.coeff_modulus().size();
    size_t n = parms.poly_modulus_degree();
    auto& key_vector = relin_keys.public_keys_;
    auto key_component_count = key_vector[0].pk_.size_;

    // Verify parameters.
    auto scheme = parms.scheme();
    auto encrypted_size = encrypted.size_;
    if (encrypted_size != 3) {
        throw invalid_argument("destination_size must be 3");
    }
    if (scheme == scheme_type::bfv && encrypted.is_ntt_form_) {
        throw invalid_argument("BFV encrypted cannot be in NTT form");
    }
    if (scheme == scheme_type::ckks && !encrypted.is_ntt_form_) {
        throw invalid_argument("CKKS encrypted must be in NTT form");
    }
    if (scheme == scheme_type::bgv && !encrypted.is_ntt_form_) {
        throw invalid_argument("BGV encrypted must be in NTT form");
    }
    if (key_component_count != 2) {
        throw invalid_argument("destination_size must be equal to key_component_count");
    }

    uint64_t* c2 = encrypted.data() + 2 * decomp_modulus_size * n;
    switch_key_inplace(context, encrypted, c2, relin_keys, true);

    // update the encrypted
    encrypted.resize(key_component_count, decomp_modulus_size, n);
}

void rescale_to_next(const PhantomContext& context, const PhantomCiphertext& encrypted,
                     PhantomCiphertext& destination) {
    auto& context_data = context.get_context_data(context.get_first_index());
    auto& parms = context_data.parms();
    auto max_chain_index = parms.coeff_modulus().size();
    auto scheme = parms.scheme();

    // Verify parameters.
    if (encrypted.chain_index() == max_chain_index) {
        throw invalid_argument("end of modulus switching chain reached");
    }

    switch (scheme) {
        case scheme_type::bfv:
            throw invalid_argument("unsupported operation for scheme type");

        case scheme_type::ckks:
            // Modulus switching with scaling
            mod_switch_scale_to_next(context, encrypted, destination);
            break;

        default:
            throw invalid_argument("unsupported scheme");
    }
}

void mod_switch_to_inplace(const PhantomContext& context, PhantomPlaintext& plain, size_t chain_index) {
    if (!plain.is_ntt_form()) {
        throw invalid_argument("plain is not in NTT form");
    }
    if (plain.chain_index() > chain_index) {
        throw invalid_argument("cannot switch to higher level modulus");
    }

    while (plain.chain_index() != chain_index) {
        mod_switch_to_next_inplace(context, plain);
    }
}

void mod_switch_to_next_inplace(const PhantomContext& context, PhantomPlaintext& plain) {
    auto& context_data = context.get_context_data(context.get_first_index());
    auto& parms = context_data.parms();
    auto coeff_modulus_size = parms.coeff_modulus().size();

    auto max_chain_index = coeff_modulus_size;
    if (!plain.is_ntt_form()) {
        throw invalid_argument("plain is not in NTT form");
    }
    if (plain.chain_index() == max_chain_index) {
        throw invalid_argument("end of modulus switching chain reached");
    }

    auto next_chain_index = plain.chain_index() + 1;
    auto& next_context_data = context.get_context_data(next_chain_index);
    auto& next_parms = next_context_data.parms();

    // q_1,...,q_{k-1}
    auto& next_coeff_modulus = next_parms.coeff_modulus();
    size_t next_coeff_modulus_size = next_coeff_modulus.size();
    size_t coeff_count = next_parms.poly_modulus_degree();

    // Compute destination size first for exception safety
    auto dest_size = next_coeff_modulus_size * coeff_count;

    Pointer<uint64_t> data_copy;
    data_copy.acquire(plain.data_);
    plain.data_.acquire(allocate<uint64_t>(global_pool(), dest_size));
    CUDA_CHECK(hipMemcpy(plain.data(), data_copy.get(), dest_size * sizeof(uint64_t), hipMemcpyDeviceToDevice));

    plain.chain_index() = next_chain_index;
}

void mod_switch_to_inplace(const PhantomContext& context, PhantomCiphertext& encrypted, size_t chain_index) {
    if (encrypted.chain_index() > chain_index) {
        throw invalid_argument("cannot switch to higher level modulus");
    }

    while (encrypted.chain_index() != chain_index) {
        mod_switch_to_next_inplace(context, encrypted);
    }
}

void mod_switch_to_next(const PhantomContext& context, const PhantomCiphertext& encrypted,
                        PhantomCiphertext& destination) {
    // Assuming at this point encrypted is already validated.
    auto& context_data = context.get_context_data(context.get_first_index());
    auto& parms = context_data.parms();
    auto coeff_modulus_size = parms.coeff_modulus().size();
    auto scheme = parms.scheme();

    auto max_chain_index = coeff_modulus_size;
    if (encrypted.chain_index() == max_chain_index) {
        throw invalid_argument("end of modulus switching chain reached");
    }
    if (parms.scheme() == scheme_type::bfv && encrypted.is_ntt_form()) {
        throw std::invalid_argument("BFV encrypted cannot be in NTT form");
    }
    if (parms.scheme() == scheme_type::ckks && !(encrypted.is_ntt_form())) {
        throw std::invalid_argument("CKKS encrypted must be in NTT form");
    }

    switch (scheme) {
        case scheme_type::bfv:
            // Modulus switching with scaling
            mod_switch_scale_to_next(context, encrypted, destination);
            break;

        case scheme_type::ckks:
            // Modulus switching without scaling
            mod_switch_drop_to_next(context, encrypted, destination);
            break;

        default:
            throw invalid_argument("unsupported scheme");
    }
}

void mod_switch_drop_to_next(const PhantomContext& context, const PhantomCiphertext& encrypted,
                             PhantomCiphertext& destination) {
    // Assuming at this point encrypted is already validated.
    auto& context_data = context.get_context_data(encrypted.chain_index());
    auto& parms = context_data.parms();
    auto coeff_modulus_size = parms.coeff_modulus().size();
    size_t N = parms.poly_modulus_degree();

    // Extract encryption parameters.
    auto next_chain_index = encrypted.chain_index() + 1;
    auto& next_context_data = context.get_context_data(next_chain_index);
    auto& next_parms = next_context_data.parms();

    // q_1,...,q_{k-1}
    size_t encrypted_size = encrypted.size();
    size_t next_coeff_modulus_size = next_parms.coeff_modulus().size();

    if (&encrypted == &destination) {
        Pointer<uint64_t> temp;
        temp.acquire(destination.data_);
        destination.data_.acquire(allocate<uint64_t>(global_pool(), encrypted_size * next_coeff_modulus_size * N));
        for (size_t i{0}; i < encrypted_size; i++) {
            auto temp_iter = temp.get() + i * coeff_modulus_size * N;
            auto encrypted_iter = encrypted.data() + i * next_coeff_modulus_size * N;
            CUDA_CHECK(hipMemcpy(encrypted_iter, temp_iter, next_coeff_modulus_size * N * sizeof(uint64_t),
                hipMemcpyDeviceToDevice));
        }
        // Set other attributes
        destination.chain_index() = next_chain_index;
        destination.coeff_modulus_size_ = next_coeff_modulus_size;
    } else {
        // Resize destination before writing
        destination.resize(context, next_chain_index, encrypted_size);
        // Copy data over to destination; only copy the RNS components relevant after modulus drop
        for (size_t i = 0; i < encrypted_size; i++) {
            auto destination_iter = destination.data() + i * next_coeff_modulus_size * N;
            auto encrypted_iter = encrypted.data() + i * coeff_modulus_size * N;
            CUDA_CHECK(hipMemcpy(destination_iter, encrypted_iter, next_coeff_modulus_size * N * sizeof(uint64_t),
                hipMemcpyDeviceToDevice));
        }
        // Set other attributes
        destination.scale() = encrypted.scale();
        destination.is_ntt_form() = encrypted.is_ntt_form();
    }
}

void mod_switch_scale_to_next(const PhantomContext& context, const PhantomCiphertext& encrypted,
                              PhantomCiphertext& destination) {
    // Assuming at this point encrypted is already validated.
    auto& context_data = context.get_context_data(encrypted.chain_index());
    auto& parms = context_data.parms();
    auto& rns_tool = context.get_context_data(encrypted.chain_index()).gpu_rns_tool();

    // Extract encryption parameters.
    size_t coeff_mod_size = parms.coeff_modulus().size();
    size_t poly_degree = parms.poly_modulus_degree();
    size_t encrypted_size = encrypted.size();

    auto next_index_id = context.get_next_index(encrypted.chain_index());
    auto& next_context_data = context.get_context_data(next_index_id);
    auto& next_parms = next_context_data.parms();

    //    size_t next_coeff_modulus_size = next_parms.coeff_modulus().size();

    Pointer<uint64_t> encrypted_copy;
    encrypted_copy.acquire(allocate<uint64_t>(global_pool(), encrypted_size * coeff_mod_size * poly_degree));
    CUDA_CHECK(hipMemcpy(encrypted_copy.get(), encrypted.data(),
        encrypted_size * coeff_mod_size * poly_degree * sizeof(uint64_t), hipMemcpyDeviceToDevice));
    // resize and empty the data array
    destination.resize(context, next_index_id, encrypted_size);

    switch (next_parms.scheme()) {
        case scheme_type::bfv:
            rns_tool.divide_and_round_q_last(encrypted_copy.get(), encrypted_size, destination.data());
            break;

        case scheme_type::ckks:
            rns_tool.divide_and_round_q_last_ntt(encrypted_copy.get(), encrypted_size, context.gpu_rns_tables(),
                                                 destination.data());
            break;

        default:
            throw invalid_argument("unsupported scheme");
    }

    // Set other attributes
    destination.is_ntt_form() = encrypted.is_ntt_form();
    if (next_parms.scheme() == scheme_type::ckks) {
        // Change the scale when using CKKS
        destination.scale() = encrypted.scale() / static_cast<double>(parms.coeff_modulus().back().value());
    }
}

void rotate_internal(const PhantomContext& context, PhantomCiphertext& encrypted, int step,
                     const PhantomGaloisKey& galois_key) {
    auto& context_data = context.get_context_data(encrypted.chain_index_);

    // Is there anything to do?
    if (step == 0) {
        return;
    }

    size_t coeff_count = context_data.parms().poly_modulus_degree();
    auto& key_galois_tool = context.key_galois_tool_;
    auto& galois_elts = key_galois_tool->galois_elts_;
    auto step_galois_elt = key_galois_tool->get_elt_from_step(step);

    auto iter = find(galois_elts.begin(), galois_elts.end(), step_galois_elt);
    if (iter != galois_elts.end()) {
        auto galois_elt_index = iter - galois_elts.begin();
        // Perform rotation and key switching
        apply_galois_inplace(context, encrypted, galois_elt_index, galois_key);
    } else {
        // Convert the steps to NAF: guarantees using smallest HW
        vector<int> naf_step = naf(step);

        // If naf_steps contains only one element, then this is a power-of-two
        // rotation and we would have expected not to get to this part of the
        // if-statement.
        if (naf_step.size() == 1) {
            throw invalid_argument("Galois key not present");
        }
        for (auto temp_step: naf_step) {
            if (static_cast<size_t>(abs(temp_step)) != (coeff_count >> 1)) {
                rotate_internal(context, encrypted, temp_step, galois_key);
            }
        }
    }
}

void hoisting_inplace(const PhantomContext& context, PhantomCiphertext& ct, const PhantomGaloisKey& glk,
                      const std::vector<int>& steps) {
    if (ct.size() > 2)
        throw invalid_argument("ciphertext size must be 2");

    auto& context_data = context.get_context_data(ct.chain_index_);
    auto& key_context_data = context.get_context_data(0);
    auto& key_parms = key_context_data.parms();
    auto scheme = key_parms.scheme();
    auto n = key_parms.poly_modulus_degree();
    auto mul_tech = key_parms.mul_tech();
    auto& key_modulus = key_parms.coeff_modulus();
    size_t size_P = key_parms.special_modulus_size();
    size_t size_QP = key_modulus.size();

    // HPS and HPSOverQ does not drop modulus
    uint32_t levelsDropped;

    if (scheme == scheme_type::bfv) {
        levelsDropped = 0;
        if (mul_tech == mul_tech_type::hps_overq_leveled) {
            size_t depth = ct.GetNoiseScaleDeg();
            bool isKeySwitch = true;
            bool is_Asymmetric = ct.is_asymmetric();
            size_t levels = depth - 1;
            auto dcrtBits = static_cast<double>(context.get_context_data(1).gpu_rns_tool().qMSB());

            // how many levels to drop
            levelsDropped = FindLevelsToDrop(context, levels, dcrtBits, isKeySwitch, is_Asymmetric);
        }
    } else if (scheme == scheme_type::bgv || scheme == scheme_type::ckks) {
        levelsDropped = ct.chain_index() - 1;
    } else {
        throw invalid_argument("unsupported scheme in switch_key_inplace");
    }

    auto& rns_tool = context.get_context_data(1 + levelsDropped).gpu_rns_tool();
    auto& parms = context_data.parms();
    auto& key_galois_tool = context.key_galois_tool_;
    auto& galois_elts = key_galois_tool->galois_elts_;

    auto modulus_QP = context.gpu_rns_tables().modulus();

    size_t size_Ql = rns_tool.base_Ql().size();
    size_t size_Q = size_QP - size_P;
    size_t size_QlP = size_Ql + size_P;

    auto size_Q_n = size_Q * n;
    auto size_Ql_n = size_Ql * n;
    auto size_QP_n = size_QP * n;
    auto size_QlP_n = size_QlP * n;

    Pointer<uint64_t> c0;
    c0.acquire(allocate<uint64_t>(global_pool(), size_Ql_n));

    Pointer<uint64_t> c1;
    c1.acquire(allocate<uint64_t>(global_pool(), size_Ql_n));

    auto elts = key_galois_tool->get_elts_from_steps(steps);

    // ------------------------------------------ automorphism c0 ------------------------------------------------------

    // specific operations for HPSOverQLeveled
    if (mul_tech == mul_tech_type::hps_overq_leveled && levelsDropped) {
        rns_tool.scaleAndRound_HPS_Q_Ql(c0.get(), ct.data());
    } else {
        CUDA_CHECK(hipMemcpy(c0.get(), ct.data(), size_Ql_n * sizeof(uint64_t), hipMemcpyDeviceToDevice));
    }

    Pointer<uint64_t> acc_c0;
    acc_c0.acquire(allocate<uint64_t>(global_pool(), size_Ql_n));

    auto first_elt = elts[0];
    auto first_iter = find(galois_elts.begin(), galois_elts.end(), first_elt);
    if (first_iter == galois_elts.end())
        throw std::logic_error("Galois key not present in hoisting");
    auto first_elt_index = first_iter - galois_elts.begin();

    if (parms.scheme() == scheme_type::bfv) {
        key_galois_tool->apply_galois(c0.get(), context.gpu_rns_tables(), size_Ql, first_elt_index, acc_c0.get());
    } else if (parms.scheme() == scheme_type::ckks || parms.scheme() == scheme_type::bgv) {
        key_galois_tool->apply_galois_ntt(c0.get(), size_Ql, first_elt_index, acc_c0.get());
    } else {
        throw logic_error("scheme not implemented");
    }

    // ----------------------------------------------- modup c1 --------------------------------------------------------

    // specific operations for HPSOverQLeveled
    if (mul_tech == mul_tech_type::hps_overq_leveled && levelsDropped) {
        rns_tool.scaleAndRound_HPS_Q_Ql(c1.get(), ct.data() + size_Q_n);
    } else {
        CUDA_CHECK(hipMemcpy(c1.get(), ct.data() + size_Ql_n, size_Ql_n * sizeof(uint64_t), hipMemcpyDeviceToDevice));
    }

    // Prepare key
    auto& key_vector = glk.relin_keys_[first_elt_index].public_keys_;
    auto key_poly_num = key_vector[0].pk_.size_;
    if (key_poly_num != 2)
        throw std::invalid_argument("key_poly_num must be 2 in hoisting");

    size_t beta = rns_tool.v_base_part_Ql_to_compl_part_QlP_conv().size();

    // mod up
    Pointer<uint64_t> modup_c1;
    modup_c1.acquire(allocate<uint64_t>(global_pool(), beta * size_QlP_n));
    rns_tool.modup(modup_c1.get(), c1.get(), context.gpu_rns_tables(), scheme);

    // ------------------------------------------ automorphism c1 ------------------------------------------------------

    Pointer<uint64_t> temp_modup_c1;
    temp_modup_c1.acquire(allocate<uint64_t>(global_pool(), beta * size_QlP_n));

    for (size_t b = 0; b < beta; b++) {
        key_galois_tool->apply_galois_ntt(modup_c1.get() + b * size_QlP_n, size_QlP, first_elt_index,
                                          temp_modup_c1.get() + b * size_QlP_n);
    }

    // ----------------------------------------- inner product c1 ------------------------------------------------------

    Pointer<uint64_t> acc_cx;
    acc_cx.acquire(allocate<uint64_t>(global_pool(), 2 * size_QlP_n));

    auto reduction_threshold =
            (1 << (bits_per_uint64 - static_cast<uint64_t>(log2(key_modulus.front().value())) - 1)) - 1;
    key_switch_inner_prod_c2_and_evk<<<size_QlP_n / blockDimGlb.x, blockDimGlb>>>(
        acc_cx.get(), temp_modup_c1.get(), glk.relin_keys_[first_elt_index].public_keys_ptr_.get(), modulus_QP, n,
        size_QP, size_QP_n, size_QlP, size_QlP_n, size_Q, size_Ql, beta, reduction_threshold);

    // ------------------------------------------ loop accumulate ------------------------------------------------------
    Pointer<uint64_t> temp_c0;
    temp_c0.acquire(allocate<uint64_t>(global_pool(), size_Ql_n));

    for (size_t i = 1; i < elts.size(); i++) {
        // automorphism c0

        auto elt = elts[i];
        auto iter = find(galois_elts.begin(), galois_elts.end(), elt);
        if (iter == galois_elts.end())
            throw std::logic_error("Galois key not present in hoisting");
        auto elt_index = iter - galois_elts.begin();

        if (parms.scheme() == scheme_type::bfv) {
            key_galois_tool->apply_galois(c0.get(), context.gpu_rns_tables(), size_Ql, elt_index, temp_c0.get());
        } else if (parms.scheme() == scheme_type::ckks || parms.scheme() == scheme_type::bgv) {
            key_galois_tool->apply_galois_ntt(c0.get(), size_Ql, elt_index, temp_c0.get());
        } else {
            throw logic_error("scheme not implemented");
        }

        // add to acc_c0
        uint64_t gridDimGlb = size_Ql_n / blockDimGlb.x;
        add_rns_poly<<<gridDimGlb, blockDimGlb>>>(acc_c0.get(), temp_c0.get(), rns_tool.base_Ql().base(), acc_c0.get(),
                                                  n, size_Ql);

        // automorphism c1

        for (size_t b = 0; b < beta; b++) {
            key_galois_tool->apply_galois_ntt(modup_c1.get() + b * size_QlP_n, size_QlP, elt_index,
                                              temp_modup_c1.get() + b * size_QlP_n);
        }

        // inner product c1

        Pointer<uint64_t> temp_cx;
        temp_cx.acquire(allocate<uint64_t>(global_pool(), 2 * size_QlP_n));

        key_switch_inner_prod_c2_and_evk<<<size_QlP_n / blockDimGlb.x, blockDimGlb>>>(
            temp_cx.get(), temp_modup_c1.get(), glk.relin_keys_[elt_index].public_keys_ptr_.get(), modulus_QP, n,
            size_QP, size_QP_n, size_QlP, size_QlP_n, size_Q, size_Ql, beta, reduction_threshold);

        // add to acc_cx
        gridDimGlb = size_QlP_n / blockDimGlb.x;
        add_rns_poly<<<gridDimGlb, blockDimGlb>>>(acc_cx.get(), temp_cx.get(), rns_tool.base_QlP().base(), acc_cx.get(),
                                                  n, size_QlP);
        add_rns_poly<<<gridDimGlb, blockDimGlb>>>(acc_cx.get() + size_QlP_n, temp_cx.get() + size_QlP_n,
                                                  rns_tool.base_QlP().base(), acc_cx.get() + size_QlP_n, n, size_QlP);
    }

    // -------------------------------------------- mod down c1 --------------------------------------------------------
    rns_tool.moddown_from_NTT(acc_cx.get(), acc_cx.get(), context.gpu_rns_tables(), scheme);
    rns_tool.moddown_from_NTT(acc_cx.get() + size_QlP_n, acc_cx.get() + size_QlP_n, context.gpu_rns_tables(), scheme);

    // new c0
    if (mul_tech == mul_tech_type::hps_overq_leveled && levelsDropped) {
        add_rns_poly<<<size_Ql_n / blockDimGlb.x, blockDimGlb>>>(acc_c0.get(), acc_cx.get(), rns_tool.base_Ql().base(),
                                                                 acc_cx.get(), n, size_Ql);
        rns_tool.ExpandCRTBasis_Ql_Q(ct.data(), acc_cx.get());
    } else {
        add_rns_poly<<<size_Ql_n / blockDimGlb.x, blockDimGlb>>>(acc_c0.get(), acc_cx.get(), rns_tool.base_Ql().base(),
                                                                 ct.data(), n, size_Ql);
    }

    // new c1
    if (mul_tech == mul_tech_type::hps_overq_leveled && levelsDropped) {
        rns_tool.ExpandCRTBasis_Ql_Q(ct.data() + size_Q_n, acc_cx.get() + size_QlP_n);
    } else {
        CUDA_CHECK(hipMemcpy(ct.data() + size_Ql_n, acc_cx.get() + size_QlP_n, size_Ql_n * sizeof(uint64_t),
            hipMemcpyDeviceToDevice));
    }
}

void apply_galois_inplace(const PhantomContext& context, PhantomCiphertext& encrypted, size_t galois_elt_index,
                          const PhantomGaloisKey& galois_keys) {
    auto& context_data = context.get_context_data(encrypted.chain_index_);
    auto& parms = context_data.parms();
    auto& coeff_modulus = parms.coeff_modulus();
    size_t N = parms.poly_modulus_degree();
    size_t coeff_modulus_size = coeff_modulus.size();
    size_t encrypted_size = encrypted.size();
    if (encrypted_size > 2) {
        throw invalid_argument("encrypted size must be 2");
    }
    auto c0 = encrypted.data();
    auto c1 = encrypted.data() + encrypted.coeff_modulus_size_ * encrypted.poly_modulus_degree_;
    // Use key_context_data where permutation tables exist since previous runs.
    auto& key_galois_tool = context.key_galois_tool_;

    Pointer<uint64_t> temp;
    temp.acquire(allocate<uint64_t>(global_pool(), coeff_modulus_size * N));

    // DO NOT CHANGE EXECUTION ORDER OF FOLLOWING SECTION
    // BEGIN: Apply Galois for each ciphertext
    // Execution order is sensitive, since apply_galois is not inplace!
    if (parms.scheme() == scheme_type::bfv) {
        // !!! DO NOT CHANGE EXECUTION ORDER!!!
        // First transform c0
        key_galois_tool->apply_galois(c0, context.gpu_rns_tables(), coeff_modulus_size, galois_elt_index, temp.get());
        // Copy result to c0
        CUDA_CHECK(hipMemcpy(c0, temp.get(), coeff_modulus_size * N * sizeof(uint64_t), hipMemcpyDeviceToDevice));
        // Next transform c1
        key_galois_tool->apply_galois(c1, context.gpu_rns_tables(), coeff_modulus_size, galois_elt_index, temp.get());
    } else if (parms.scheme() == scheme_type::ckks || parms.scheme() == scheme_type::bgv) {
        // !!! DO NOT CHANGE EXECUTION ORDER!!
        // First transform c0
        key_galois_tool->apply_galois_ntt(c0, coeff_modulus_size, galois_elt_index, temp.get());
        // Copy result to c0
        CUDA_CHECK(hipMemcpy(c0, temp.get(), coeff_modulus_size * N * sizeof(uint64_t), hipMemcpyDeviceToDevice));
        // Next transform c1
        key_galois_tool->apply_galois_ntt(c1, coeff_modulus_size, galois_elt_index, temp.get());
    } else {
        throw logic_error("scheme not implemented");
    }

    // Wipe c1
    CUDA_CHECK(hipMemset(c1, 0, coeff_modulus_size * N * sizeof(uint64_t)));

    // END: Apply Galois for each ciphertext
    // REORDERING IS SAFE NOW
    // Calculate (temp * galois_key[0], temp * galois_key[1]) + (c0, 0)
    switch_key_inplace(context, encrypted, temp.get(), galois_keys.relin_keys_[galois_elt_index]);
}

void conjugate_internal(const PhantomContext& context, PhantomCiphertext& encrypted,
                        const PhantomGaloisKey& galois_key) {
    constexpr size_t galois_elt_index = 0;
    apply_galois_inplace(context, encrypted, galois_elt_index, galois_key);
}

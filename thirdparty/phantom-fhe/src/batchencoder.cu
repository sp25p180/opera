#include "hip/hip_runtime.h"
#include "batchencoder.h"

using namespace std;
using namespace phantom;
using namespace phantom::util;

PhantomBatchEncoder::PhantomBatchEncoder(const PhantomContext &context) {
    auto &context_data = context.get_context_data(0);
    auto &parms = context_data.parms();
    if (parms.scheme() != scheme_type::bfv && parms.scheme() != scheme_type::bgv) {
        throw std::invalid_argument("unsupported scheme");
    }

    // Set the slot count
    auto poly_degree = parms.poly_modulus_degree();
    slots_ = poly_degree;

    // Populate matrix representation index map
    data_.acquire(allocate<uint64_t>(global_pool(), slots_));
    matrix_reps_index_map_.acquire(allocate<uint64_t>(global_pool(), slots_));
    populate_matrix_reps_index_map();
}

void PhantomBatchEncoder::populate_matrix_reps_index_map() const {
    vector<uint64_t> temp;
    int logn = get_power_of_two(slots_);
    // Copy from the matrix to the value vectors
    size_t row_size = slots_ >> 1;
    size_t m = slots_ << 1;
    uint64_t gen = 5;
    uint64_t pos = 1;
    temp.resize(slots_);
    for (size_t i = 0; i < row_size; i++) {
        // Position in normal bit order
        uint64_t index1 = (pos - 1) >> 1;
        uint64_t index2 = (m - pos - 1) >> 1;

        // Set the bit-reversed locations
        temp[i] = (uint64_t)(util::reverse_bits(index1, logn));
        temp[row_size | i] = static_cast<size_t>(util::reverse_bits(index2, logn));

        // Next primitive root
        pos *= gen;
        pos &= (m - 1);
    }
    CUDA_CHECK(
            hipMemcpy(matrix_reps_index_map_.get(), temp.data(), sizeof(uint64_t) * slots_, hipMemcpyHostToDevice));
}

__global__ void encode_gpu(uint64_t *out, uint64_t *in, size_t in_size, uint64_t *index_map, uint64_t mod,
                           size_t slots) {
    for (size_t tid = blockIdx.x * blockDim.x + threadIdx.x; tid < slots; tid += blockDim.x * gridDim.x) {
        if (tid < in_size) {
            const uint64_t temp = in[tid];
            out[index_map[tid]] = temp + (temp >> 63) * mod;
        }
        else
            out[index_map[tid]] = 0;
    }
}

// TODO: support <uint64_t> type
void PhantomBatchEncoder::encode(const PhantomContext &context, const std::vector<int64_t> &values_matrix,
                                 PhantomPlaintext &destination) {
    auto &context_data = context.get_context_data(0);
    auto &parms = context_data.parms();
    auto &plain_modulus = parms.plain_modulus();
    size_t values_matrix_size = values_matrix.size();
    if (values_matrix_size > slots_) {
        throw std::logic_error("values_matrix size is too large");
    }

    CUDA_CHECK(hipMemcpy(data_.get(), values_matrix.data(), values_matrix.size() * sizeof(uint64_t),
                          hipMemcpyHostToDevice));

    uint64_t gridDimGlb = ceil(slots_ / blockDimGlb.x);
    encode_gpu<<<gridDimGlb, blockDimGlb>>>(destination.data(), data_.get(), values_matrix_size,
                                            matrix_reps_index_map_.get(), plain_modulus.value(), slots_);

    nwt_2d_radix8_backward_inplace(destination.data(), context.gpu_plain_tables(), 1, 0);
}

__global__ void decode_gpu(uint64_t *out, uint64_t *in, uint64_t *index_map, uint64_t slots) {
    for (size_t tid = blockIdx.x * blockDim.x + threadIdx.x; tid < slots; tid += blockDim.x * gridDim.x) {
        out[tid] = in[index_map[tid]];
    }
}

void PhantomBatchEncoder::decode(const PhantomContext &context, const PhantomPlaintext &plain,
                                 std::vector<std::int64_t> &destination) const {
    if (plain.is_ntt_form()) {
        throw std::invalid_argument("plain cannot be in NTT form");
    }

    destination.resize(plain.poly_modulus_degree_);

    // Copy plain.data_
    Pointer<uint64_t> plain_data_copy;
    plain_data_copy.acquire(allocate<uint64_t>(global_pool(), slots_));
    CUDA_CHECK(hipMemcpy(plain_data_copy.get(), plain.data(), slots_ * sizeof(uint64_t), hipMemcpyDeviceToDevice));

    nwt_2d_radix8_forward_inplace(plain_data_copy.get(), context.gpu_plain_tables(), 1, 0);

    Pointer<uint64_t> out;
    out.acquire(allocate<uint64_t>(global_pool(), slots_));
    uint64_t gridDimGlb = ceil(slots_ / blockDimGlb.x);
    decode_gpu<<<gridDimGlb, blockDimGlb>>>(out.get(), plain_data_copy.get(), matrix_reps_index_map_.get(), slots_);

    CUDA_CHECK(hipMemcpy(destination.data(), out.get(), sizeof(uint64_t) * slots_, hipMemcpyDeviceToHost));
}

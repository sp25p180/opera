#include "hip/hip_runtime.h"
// Copyright (c) Microsoft Corporation. All rights reserved.
// Licensed under the MIT license.

#include "util/common.h"
#include "util/uintarith.h"
#include "util/uintcore.h"
#include <algorithm>
#include <string>

using namespace std;

namespace phantom {
    namespace util {
        string uint_to_hex_string(const uint64_t *value, size_t uint64_count) {
            // Start with a string with a zero for each nibble in the array.
            size_t num_nibbles = mul_safe(uint64_count, static_cast<size_t>(nibbles_per_uint64));
            string output(num_nibbles, '0');

            // Iterate through each uint64 in array and set string with correct nibbles in hex.
            size_t nibble_index = num_nibbles;
            size_t leftmost_non_zero_pos = num_nibbles;
            for (size_t i = 0; i < uint64_count; i++) {
                uint64_t part = *value++;

                // Iterate through each nibble in the current uint64.
                for (size_t j = 0; j < nibbles_per_uint64; j++) {
                    size_t nibble = (size_t)(part & uint64_t(0x0F));
                    size_t pos = --nibble_index;
                    if (nibble != 0) {
                        // If nibble is not zero, then update string and save this pos to determine
                        // number of leading zeros.
                        output[pos] = nibble_to_upper_hex(static_cast<int>(nibble));
                        leftmost_non_zero_pos = pos;
                    }
                    part >>= 4;
                }
            }

            // Trim string to remove leading zeros.
            output.erase(0, leftmost_non_zero_pos);

            // Return 0 if nothing remains.
            if (output.empty()) {
                return string("0");
            }

            return output;
        }

        string uint_to_dec_string(const uint64_t *value, size_t uint64_count) {
            if (!uint64_count) {
                return string("0");
            }
            auto remainder = std::vector<uint64_t>(uint64_count);
            auto quotient = std::vector<uint64_t>(uint64_count);
            auto base = std::vector<uint64_t>(uint64_count);

            uint64_t *remainderptr = remainder.data();
            uint64_t *quotientptr = quotient.data();
            uint64_t *baseptr = base.data();
            set_uint(10, uint64_count, baseptr);
            set_uint(value, uint64_count, remainderptr);
            string output;
            while (!is_zero_uint(remainderptr, uint64_count)) {
                divide_uint_inplace(remainderptr, baseptr, uint64_count, quotientptr);
                char digit = (char)(remainderptr[0] + uint64_t('0'));
                output += digit;
                swap(remainderptr, quotientptr);
            }
            reverse(output.begin(), output.end());

            // Return 0 if nothing remains.
            if (output.empty()) {
                return string("0");
            }

            return output;
        }
    }
}

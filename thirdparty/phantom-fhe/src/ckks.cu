#include "hip/hip_runtime.h"
#include "ckks.h"

using namespace std;
using namespace phantom;
using namespace phantom::util;
using namespace phantom::arith;

__global__ void bit_reverse_and_zero_padding(hipDoubleComplex* dst, hipDoubleComplex* src, uint64_t in_size,
                                             uint32_t slots, uint32_t logn) {
    for (uint32_t tid = blockIdx.x * blockDim.x + threadIdx.x;
         tid < slots;
         tid += blockDim.x * gridDim.x) {
        if (tid < uint32_t(in_size)) {
            dst[reverse_bits_uint32(tid, logn)] = src[tid];
        } else {
            dst[reverse_bits_uint32(tid, logn)] = (hipDoubleComplex){0.0, 0.0};
        }
    }
}

__global__ void bit_reverse(hipDoubleComplex* dst, hipDoubleComplex* src, uint32_t slots, uint32_t logn) {
    for (uint32_t tid = blockIdx.x * blockDim.x + threadIdx.x;
         tid < slots;
         tid += blockDim.x * gridDim.x) {
        dst[reverse_bits_uint32(tid, logn)] = src[tid];
    }
}

PhantomCKKSEncoder::PhantomCKKSEncoder(const PhantomContext& context) // : context_(context)
{
    auto& context_data = context.get_context_data(first_chain_index_);
    auto& parms = context_data.parms();
    auto& coeff_modulus = parms.coeff_modulus();
    std::size_t coeff_modulus_size = coeff_modulus.size();
    std::size_t coeff_count = parms.poly_modulus_degree();

    if (parms.scheme() != scheme_type::ckks) {
        throw std::invalid_argument("unsupported scheme");
    }
    uint32_t logn = get_power_of_two(coeff_count);
    slots_ = coeff_count >> 1; // n/2
    uint32_t m = coeff_count << 1;
    uint32_t slots_half = slots_ >> 1;
    gpu_ckks_msg_vec_ = DCKKSEncoderInfo(coeff_count);

    // We need m powers of the primitive 2n-th root, m = 2n
    root_powers_.reserve(m);
    rotation_group_.reserve(slots_half);

    uint32_t gen = 5;
    uint32_t pos = 1; // Position in normal bit order
    for (size_t i = 0; i < slots_half; i++) {
        // Set the bit-reversed locations
        rotation_group_[i] = pos;

        // Next primitive root
        pos *= gen; // 5^i mod m
        pos &= (m - 1);
    }

    // Powers of the primitive 2n-th root have 4-fold symmetry
    if (m >= 8) {
        complex_roots_ = make_shared<util::ComplexRoots>(util::ComplexRoots(static_cast<size_t>(m)));
        for (size_t i = 0; i < m; i++) {
            root_powers_[i] = complex_roots_->get_root(i);
        }
    } else if (m == 4) {
        root_powers_[0] = {1, 0};
        root_powers_[1] = {0, 1};
        root_powers_[2] = {-1, 0};
        root_powers_[3] = {0, -1};
    }

    CUDA_CHECK(
        hipMemcpy(gpu_ckks_msg_vec_.twiddle(), root_powers_.data(), m * sizeof(hipDoubleComplex), hipMemcpyHostToDevice
        ));
    CUDA_CHECK(
        hipMemcpy(gpu_ckks_msg_vec_.mul_group(), rotation_group_.data(), slots_half * sizeof(uint32_t),
            hipMemcpyHostToDevice));

    // CUDA_CHECK(hipStreamAttachMemAsync(NULL, gpu_ckks_msg_vec_.twiddle(), 0, hipMemAttachGlobal));
    // CUDA_CHECK(hipStreamAttachMemAsync(NULL, gpu_ckks_msg_vec_.mul_group(), 0, hipMemAttachGlobal));

    // Create gpu_plain_rns
    // auto &small_ntt_tables = context.get_context_data(0).small_ntt_tables();
}

void PhantomCKKSEncoder::encode_internal(const PhantomContext& context, const hipDoubleComplex* values,
                                         size_t values_size, size_t chain_index, double scale,
                                         PhantomPlaintext& destination) {
    auto& context_data = context.get_context_data(chain_index);
    auto& parms = context_data.parms();
    auto& coeff_modulus = parms.coeff_modulus();
    auto& rns_tool = context_data.gpu_rns_tool();
    std::size_t coeff_modulus_size = coeff_modulus.size();
    std::size_t coeff_count = parms.poly_modulus_degree();

    if (!values && values_size > 0) {
        throw std::invalid_argument("values cannot be null");
    }
    if (values_size > slots_) {
        throw std::invalid_argument("values_size is too large");
    }

    // CUDA_CHECK(hipMallocManaged((void **)&(destination.data_), coeff_count * coeff_modulus_size * sizeof(uint64_t)));

    // Check that scale is positive and not too large
    if (scale <= 0 || (static_cast<int>(log2(scale)) + 1 >= context_data.total_coeff_modulus_bit_count())) {
        throw std::invalid_argument("scale out of bounds");
    }

    if (sparse_slots_ == 0) {
        uint32_t log_sparse_slots = ceil(log2(values_size));
        sparse_slots_ = 1 << log_sparse_slots;
    } else {
        if (values_size > sparse_slots_) {
            throw std::invalid_argument("values_size exceeds previous message length");
        }
    }
    // size_t log_sparse_slots = ceil(log2(slots_));
    // sparse_slots_ = slots_;
    if (sparse_slots_ < 2) {
        throw std::invalid_argument("single value encoding is not available");
    }

    gpu_ckks_msg_vec_.set_sparse_slots(sparse_slots_);
    CUDA_CHECK(hipMemset(gpu_ckks_msg_vec_.in(), 0, slots_ * sizeof(hipDoubleComplex)));
    Pointer<hipDoubleComplex> temp;
    temp.acquire(allocate<hipDoubleComplex>(global_pool(), values_size));
    CUDA_CHECK(hipMemset(temp.get(), 0, values_size * sizeof(hipDoubleComplex)));
    CUDA_CHECK(hipMemcpy(temp.get(), values, sizeof(hipDoubleComplex) * values_size, hipMemcpyHostToDevice));

    uint32_t log_sparse_n = log2(sparse_slots_);
    uint64_t gridDimGlb = ceil(sparse_slots_ / blockDimGlb.x);
    bit_reverse_and_zero_padding<<<gridDimGlb, blockDimGlb>>>(gpu_ckks_msg_vec_.in(), temp.get(), values_size,
                                                              sparse_slots_, log_sparse_n);

    double fix = scale / static_cast<double>(sparse_slots_);

    special_fft_backward(&gpu_ckks_msg_vec_, (uint32_t) 1, fix);
    // we calculate max_coeff_bit_count at cpu side
    // CUDA_CHECK(hipStreamAttachMemAsync(NULL, gpu_ckks_msg_vec_.in(), 0, hipMemAttachGlobal));

    // TODO to opt this
    vector<hipDoubleComplex> temp2(sparse_slots_);
    CUDA_CHECK(
        hipMemcpy(temp2.data(), gpu_ckks_msg_vec_.in(), sparse_slots_ * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost
        ));

    double max_coeff = 0;
    for (std::size_t i = 0; i < sparse_slots_; i++) {
        max_coeff = std::max(max_coeff, std::fabs(temp2[i].x));
    }
    for (std::size_t i = 0; i < sparse_slots_; i++) {
        max_coeff = std::max(max_coeff, std::fabs(temp2[i].y));
    }
    // Verify that the values are not too large to fit in coeff_modulus
    // Note that we have an extra + 1 for the sign bit
    // Don't compute logarithmis of numbers less than 1
    int max_coeff_bit_count = static_cast<int>(std::ceil(std::log2(std::max(max_coeff, 1.0)))) + 1;

    if (max_coeff_bit_count >= context_data.total_coeff_modulus_bit_count()) {
        throw std::invalid_argument("encoded values are too large");
    }
    // Resize destination to appropriate size
    // Need to first set parms_id to zero, otherwise resize
    // will throw an exception.
    destination.chain_index() = 0;
    // destination.resize(util::mul_safe(coeff_count, coeff_modulus_size));
    destination.resize(coeff_modulus_size, coeff_count);

    // we can in fact find all coeff_modulus in DNTTTable structure....
    rns_tool.base_Ql().decompose_array(destination.data(), gpu_ckks_msg_vec_.in(), sparse_slots_ << 1,
                                       (uint32_t) slots_ / sparse_slots_, max_coeff_bit_count);
    // CUDA_CHECK(hipStreamAttachMemAsync(NULL, destination.data(), 0, hipMemAttachGlobal));

    nwt_2d_radix8_forward_inplace(destination.data(), context.gpu_rns_tables(), coeff_modulus_size, 0);

    destination.chain_index() = chain_index;
    destination.scale() = scale;
}

void PhantomCKKSEncoder::encode_internal(const PhantomContext& context, double value, size_t chain_index, double scale,
                                         PhantomPlaintext& destination) {
    auto& context_data = context.get_context_data(chain_index);
    auto& parms = context_data.parms();
    auto& coeff_modulus = parms.coeff_modulus();
    auto& rns_tool = context_data.gpu_rns_tool();
    const std::size_t coeff_modulus_size = coeff_modulus.size();
    const std::size_t coeff_count = parms.poly_modulus_degree();

    // CUDA_CHECK(hipMallocManaged((void **)&(destination.data_), coeff_count * coeff_modulus_size * sizeof(uint64_t)));

    // Check that scale is positive and not too large
    if (scale <= 0 || (static_cast<int>(log2(scale)) + 1 >= context_data.total_coeff_modulus_bit_count())) {
        throw std::invalid_argument("scale out of bounds");
    }

    if (sparse_slots_ == 0) {
        sparse_slots_ = slots_;
    }

    // Compute the scaled value
    value *= scale;

    int coeff_bit_count = static_cast<int>(log2(fabs(value))) + 2;
    if (coeff_bit_count >= context_data.total_coeff_modulus_bit_count()) {
        throw invalid_argument("encoded value is too large");
    }

    // Resize destination to appropriate size
    // Need to first set parms_id to zero, otherwise resize
    // will throw an exception.
    destination.chain_index() = 0;
    destination.resize(coeff_modulus_size, coeff_count);

    // decompose and fill
    rns_tool.base_Ql().decompose(destination.data(), value, coeff_count, coeff_bit_count);

    destination.chain_index() = chain_index;
    destination.scale() = scale;
}

void PhantomCKKSEncoder::encode_internal(const PhantomContext& context, int64_t value, size_t chain_index,
                                         PhantomPlaintext& destination) {
    auto& context_data = context.get_context_data(chain_index);
    auto& parms = context_data.parms();
    auto& coeff_modulus = parms.coeff_modulus();
    auto& rns_tool = context_data.gpu_rns_tool();
    const std::size_t coeff_modulus_size = coeff_modulus.size();
    const std::size_t coeff_count = parms.poly_modulus_degree();

    if (sparse_slots_ == 0) {
        sparse_slots_ = slots_;
    }

    int coeff_bit_count = static_cast<int>(log2(fabs(value))) + 2;
    if (coeff_bit_count >= context_data.total_coeff_modulus_bit_count()) {
        throw invalid_argument("encoded value is too large");
    }

    // Resize destination to appropriate size
    // Need to first set parms_id to zero, otherwise resize
    // will throw an exception.
    destination.chain_index() = 0;
    destination.resize(coeff_modulus_size, coeff_count);

    // decompose and fill
    rns_tool.base_Ql().decompose(destination.data(), value, coeff_count, coeff_bit_count);

    destination.chain_index() = chain_index;
    destination.scale() = 1.0;
}

void PhantomCKKSEncoder::decode_internal(const PhantomContext& context, const PhantomPlaintext& plain,
                                         hipDoubleComplex* destination) {
    if (!plain.is_ntt_form()) {
        throw std::invalid_argument("plain is not in NTT form");
    }
    if (!destination) {
        throw std::invalid_argument("destination cannot be null");
    }

    auto& context_data = context.get_context_data(plain.chain_index_);
    auto& parms = context_data.parms();
    auto& coeff_modulus = parms.coeff_modulus();
    auto& rns_tool = context_data.gpu_rns_tool();
    const std::size_t coeff_modulus_size = coeff_modulus.size();
    const std::size_t coeff_count = parms.poly_modulus_degree();
    const std::size_t rns_poly_uint64_count = coeff_count * coeff_modulus_size;

    // cout << endl << "chain_index = " << plain.chain_index_ << endl;

    // CUDA_CHECK(hipMallocManaged((void **)&(plain.data_), rns_poly_uint64_count * sizeof(uint64_t)));

    if (plain.scale() <= 0 ||
        (static_cast<int>(log2(plain.scale())) >= context_data.total_coeff_modulus_bit_count())) {
        throw std::invalid_argument("scale out of bounds");
    }

    auto decryption_modulus = context_data.total_coeff_modulus();
    auto upper_half_threshold = context_data.upper_half_threshold();
    int logn = util::get_power_of_two(coeff_count);
    Pointer<uint64_t> gpu_upper_half_threshold;
    gpu_upper_half_threshold.acquire(allocate<uint64_t>(global_pool(), upper_half_threshold.size()));
    CUDA_CHECK(
        hipMemcpy(gpu_upper_half_threshold.get(), upper_half_threshold.data(), upper_half_threshold.size() * sizeof(
            uint64_t), hipMemcpyHostToDevice));

    gpu_ckks_msg_vec_.set_sparse_slots(sparse_slots_);
    CUDA_CHECK(hipMemset(gpu_ckks_msg_vec_.in(), 0, slots_ * sizeof(hipDoubleComplex)));

    // Quick sanity check
    if ((logn < 0) || (coeff_count < POLY_MOD_DEGREE_MIN) || (coeff_count > POLY_MOD_DEGREE_MAX)) {
        throw std::logic_error("invalid parameters");
    }

    double inv_scale = double(1.0) / plain.scale();
    // Create mutable copy of input
    Pointer<uint64_t> plain_copy;
    plain_copy.acquire(allocate<uint64_t>(global_pool(), rns_poly_uint64_count));
    CUDA_CHECK(
        hipMemcpy(plain_copy.get(), plain.data(), rns_poly_uint64_count * sizeof(uint64_t), hipMemcpyDeviceToDevice));

    nwt_2d_radix8_backward_inplace(plain_copy.get(), context.gpu_rns_tables(), coeff_modulus_size, 0);

    // CRT-compose the polynomial
    rns_tool.base_Ql().compose_array(gpu_ckks_msg_vec().in(), plain_copy.get(), gpu_upper_half_threshold.get(),
                                     inv_scale, coeff_count, sparse_slots_ << 1, slots_ / sparse_slots_);

    special_fft_forward(&gpu_ckks_msg_vec_, (uint32_t) 1);
    // CUDA_CHECK(hipStreamAttachMemAsync(NULL, gpu_ckks_msg_vec_.in(), 0, hipMemAttachGlobal));

    // finally, bit-reverse and output
    Pointer<hipDoubleComplex> out;
    out.acquire(allocate<hipDoubleComplex>(global_pool(), sparse_slots_));
    uint32_t log_sparse_n = log2(sparse_slots_);
    uint64_t gridDimGlb = ceil(sparse_slots_ / blockDimGlb.x);
    bit_reverse<<<gridDimGlb, blockDimGlb>>>(out.get(), gpu_ckks_msg_vec_.in(), sparse_slots_, log_sparse_n);
    CUDA_CHECK(hipMemcpy(destination, out.get(), sparse_slots_ * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost));
}
